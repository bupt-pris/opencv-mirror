#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <opencv2/gpu/device/lbp.hpp>

namespace cv { namespace gpu { namespace device
{
    namespace lbp
    {
        __global__ void lbp_classify(const DevMem2D_< ::cv::gpu::device::Stage> stages, const DevMem2Di trees, const DevMem2D_< ::cv::gpu::device::ClNode> nodes,
            const DevMem2Df leaves, const DevMem2Di subsets,
            const DevMem2D_<uchar4> features, const DevMem2Di integral, float step, int subsetSize, DevMem2D_<int4> objects, float scale, int clWidth, int clHeight)
        {
            unsigned int x = threadIdx.x * step;
            unsigned int y = blockIdx.x  * step;
            int nodeOfs = 0, leafOfs = 0;
            ::cv::gpu::device::Feature evaluator;

            for (int s = 0; s < stages.cols; s++ )
            {
                ::cv::gpu::device::Stage stage = stages(0, s);
                int sum = 0;
                for (int w = 0; w < stage.ntrees; w++)
                {
                    ::cv::gpu::device::ClNode node = nodes(0, nodeOfs);
                    uchar4 feature = features(0, node.featureIdx);

                    uchar c = evaluator(y, x, feature, integral);
                    const int subsetIdx = (nodeOfs * subsetSize);
                    int idx = subsetIdx + ((c >> 5) & ( 1 << (c & 31)) ? leafOfs : leafOfs + 1);
                    sum += leaves(0, subsets(0, idx) );
                    nodeOfs++;
                    leafOfs += 2;
                }

                if (sum < stage.threshold)
                    return;
            }
            int4 rect;
            rect.x = roundf(x * scale);
            rect.y = roundf(y * scale);
            rect.z = roundf(clWidth * scale);
            rect.w = roundf(clHeight * scale);
            objects(blockIdx.x, threadIdx.x) = rect;
        }

        void cascadeClassify(const DevMem2Db bstages, const DevMem2Di trees, const DevMem2Db bnodes, const DevMem2Df leaves, const DevMem2Di subsets, const DevMem2Db bfeatures,
            const DevMem2Di integral, int workWidth, int workHeight, int clWidth, int clHeight, float scale, int step, int subsetSize, DevMem2D_<int4> objects, int minNeighbors, hipStream_t stream)
        {
            printf("CascadeClassify");
            int blocks = ceilf(workHeight / (float)step);
            int threads = ceilf(workWidth / (float)step);
            DevMem2D_< ::cv::gpu::device::Stage> stages = DevMem2D_< ::cv::gpu::device::Stage>(bstages);
            DevMem2D_<uchar4> features = (DevMem2D_<uchar4>)bfeatures;
            DevMem2D_< ::cv::gpu::device::ClNode> nodes = DevMem2D_< ::cv::gpu::device::ClNode>(bnodes);

            lbp_classify<<<blocks, threads>>>(stages, trees, nodes, leaves, subsets, features, integral, step, subsetSize, objects, scale, clWidth, clHeight);
        }
    }
}}}