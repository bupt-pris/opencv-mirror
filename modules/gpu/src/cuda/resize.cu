#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/filters.hpp"
# include <cfloat>

namespace cv { namespace gpu { namespace device
{
    namespace imgproc
    {
        template <typename Ptr2D, typename T> __global__ void resize(const Ptr2D src, float fx, float fy, DevMem2D_<T> dst)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                const float xcoo = x * fx;
                const float ycoo = y * fy;

                dst(y, x) = saturate_cast<T>(src(ycoo, xcoo));
            }
        }

        template <typename Ptr2D, typename T> __global__ void resize_area(const Ptr2D src, float fx, float fy, DevMem2D_<T> dst)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                dst(y, x) = saturate_cast<T>(src(y, x));
            }
        }

        template <template <typename> class Filter, typename T> struct ResizeDispatcherStream
        {
            static void call(DevMem2D_<T> src, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream)
            {
                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                BrdReplicate<T> brd(src.rows, src.cols);
                BorderReader< PtrStep<T>, BrdReplicate<T> > brdSrc(src, brd);
                Filter< BorderReader< PtrStep<T>, BrdReplicate<T> > > filteredSrc(brdSrc, fx, fy);

                resize<<<grid, block, 0, stream>>>(filteredSrc, fx, fy, dst);
                cudaSafeCall( hipGetLastError() );
            }
        };

        template <typename T> struct ResizeDispatcherStream<AreaFilter, T>
        {
            static void call(DevMem2D_<T> src, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream)
            {
                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                BrdConstant<T> brd(src.rows, src.cols);
                BorderReader< PtrStep<T>, BrdConstant<T> > brdSrc(src, brd);
                AreaFilter< BorderReader< PtrStep<T>, BrdConstant<T> > > filteredSrc(brdSrc, fx, fy);
                resize_area<<<grid, block, 0, stream>>>(filteredSrc, fx, fy, dst);
                cudaSafeCall( hipGetLastError() );
                if (stream == 0)
                    cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        template <typename T> struct ResizeDispatcherStream<IntegerAreaFilter, T>
        {
            static void call(DevMem2D_<T> src, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream)
            {
                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));
                BrdConstant<T> brd(src.rows, src.cols);
                BorderReader< PtrStep<T>, BrdConstant<T> > brdSrc(src, brd);
                IntegerAreaFilter< BorderReader< PtrStep<T>, BrdConstant<T> > > filteredSrc(brdSrc, fx, fy);
                resize_area<<<grid, block, 0, stream>>>(filteredSrc, fx, fy, dst);
                cudaSafeCall( hipGetLastError() );
                if (stream == 0)
                    cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        template <template <typename> class Filter, typename T> struct ResizeDispatcherNonStream
        {
            static void call(DevMem2D_<T> src, DevMem2D_<T> srcWhole, int xoff, int yoff, float fx, float fy, DevMem2D_<T> dst)
            {
                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                BrdReplicate<T> brd(src.rows, src.cols);
                BorderReader< PtrStep<T>, BrdReplicate<T> > brdSrc(src, brd);
                Filter< BorderReader< PtrStep<T>, BrdReplicate<T> > > filteredSrc(brdSrc);

                resize<<<grid, block>>>(filteredSrc, fx, fy, dst);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        #define OPENCV_GPU_IMPLEMENT_RESIZE_TEX(type) \
            texture< type , hipTextureType2D> tex_resize_ ## type (0, hipFilterModePoint, hipAddressModeClamp); \
            struct tex_resize_ ## type ## _reader \
            { \
                typedef type elem_type; \
                typedef int index_type; \
                const int xoff; \
                const int yoff; \
                __host__ tex_resize_ ## type ## _reader(int xoff_, int yoff_) : xoff(xoff_), yoff(yoff_) {} \
                __device__ __forceinline__ elem_type operator ()(index_type y, index_type x) const \
                { \
                    return tex2D(tex_resize_ ## type, x + xoff, y + yoff); \
                } \
            }; \
            template <template <typename> class Filter> struct ResizeDispatcherNonStream<Filter, type > \
            { \
                static void call(DevMem2D_< type > src, DevMem2D_< type > srcWhole, int xoff, int yoff, float fx, float fy, DevMem2D_< type > dst) \
                { \
                    dim3 block(32, 8); \
                    dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y)); \
                    bindTexture(&tex_resize_ ## type, srcWhole); \
                    tex_resize_ ## type ## _reader texSrc(xoff, yoff); \
                    if (srcWhole.cols == src.cols && srcWhole.rows == src.rows) \
                    { \
                        Filter<tex_resize_ ## type ## _reader> filteredSrc(texSrc); \
                        resize<<<grid, block>>>(filteredSrc, fx, fy, dst); \
                    } \
                    else \
                    { \
                        BrdReplicate< type > brd(src.rows, src.cols); \
                        BorderReader<tex_resize_ ## type ## _reader, BrdReplicate< type > > brdSrc(texSrc, brd); \
                        Filter< BorderReader<tex_resize_ ## type ## _reader, BrdReplicate< type > > > filteredSrc(brdSrc); \
                        resize<<<grid, block>>>(filteredSrc, fx, fy, dst); \
                    } \
                    cudaSafeCall( hipGetLastError() ); \
                    cudaSafeCall( hipDeviceSynchronize() ); \
                } \
            };

        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(uchar)
        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(uchar4)

        //OPENCV_GPU_IMPLEMENT_RESIZE_TEX(schar)
        //OPENCV_GPU_IMPLEMENT_RESIZE_TEX(char4)

        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(ushort)
        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(ushort4)

        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(short)
        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(short4)

        //OPENCV_GPU_IMPLEMENT_RESIZE_TEX(int)
        //OPENCV_GPU_IMPLEMENT_RESIZE_TEX(int4)

        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(float)
        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(float4)

        #undef OPENCV_GPU_IMPLEMENT_RESIZE_TEX

        template <template <typename> class Filter, typename T> struct ResizeDispatcher
        {
            static void call(DevMem2D_<T> src, DevMem2D_<T> srcWhole, int xoff, int yoff, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream)
            {
                if (stream == 0)
                    ResizeDispatcherNonStream<Filter, T>::call(src, srcWhole, xoff, yoff, fx, fy, dst);
                else
                    ResizeDispatcherStream<Filter, T>::call(src, fx, fy, dst, stream);
            }
        };

        template <typename T> struct ResizeDispatcher<AreaFilter, T>
        {
            static void call(DevMem2D_<T> src, DevMem2D_<T> srcWhole, int xoff, int yoff, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream)
            {
                int iscale_x = round(fx);
                int iscale_y = round(fy);

                if( std::abs(fx - iscale_x) < FLT_MIN && std::abs(fy - iscale_y) < FLT_MIN)
                    ResizeDispatcherStream<IntegerAreaFilter, T>::call(src, fx, fy, dst, stream);
                else
                    ResizeDispatcherStream<AreaFilter, T>::call(src, fx, fy, dst, stream);
            }
        };

        template <typename T> void resize_gpu(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, 
            DevMem2Db dst, int interpolation, hipStream_t stream)
        {
            typedef void (*caller_t)(DevMem2D_<T> src, DevMem2D_<T> srcWhole, int xoff, int yoff, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream);

            static const caller_t callers[4] =
            {
                ResizeDispatcher<PointFilter, T>::call,
                ResizeDispatcher<LinearFilter, T>::call,
                ResizeDispatcher<CubicFilter, T>::call,
                ResizeDispatcher<AreaFilter, T>::call
            };
            // chenge to linear if area interpolation upscaling
            if (interpolation == 3 && (fx <= 1.f || fy <= 1.f))
                interpolation = 1;

            callers[interpolation](static_cast< DevMem2D_<T> >(src), static_cast< DevMem2D_<T> >(srcWhole), xoff, yoff, fx, fy, 
                static_cast< DevMem2D_<T> >(dst), stream);
        }

        template void resize_gpu<uchar >(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<uchar2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<uchar3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<uchar4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        //template void resize_gpu<schar>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<char2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<char3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<char4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        template void resize_gpu<ushort >(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<ushort2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<ushort3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<ushort4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        template void resize_gpu<short >(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<short2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<short3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<short4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        //template void resize_gpu<int >(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<int2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<int3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<int4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        template void resize_gpu<float >(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<float2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<float3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<float4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        template<typename T> struct scan_traits{};

        template<> struct scan_traits<uchar>
        {
            typedef float scan_line_type;
        };

//        template <typename T>
//        __global__ void resize_area_scan(const DevMem2D_<T> src, DevMem2D_<T> dst, int fx, int fy,  DevMem2D_<T> buffer)
//        {
//            typedef typename scan_traits<T>::scan_line_type W;
//            extern __shared__ W line[];

//            const int x = threadIdx.x;
//            const int y = blockIdx.x;

//            if (y >= src.rows) return;

//            int offset = 1;

//            line[2 * x + 0] = src(y, 2 * x + 0);
//            line[2 * x + 1] = src(y, 2 * x + 1);

//            __syncthreads();//???
//            // reduction
//            for (int d = blockDim.x; d > 0; d >>= 1)
//            {
//                __syncthreads();
//                if (x < d)
//                {
//                    int ai = 2 * x * offset -1 + 1 * offset;
//                    int bi = 2 * x * offset -1 + 2 * offset;
//                    line[bi] += line[ai];
//                }

//                offset *= 2;
//            }

//            __syncthreads();
//            // convolution
//            if (x == 0) { line[(blockDim.x << 1) - 1] = 0; printf("offset: %d!!!!!!!!!!!!!\n", fx);}

//            for (int d = 1; d < (blockDim.x << 1); d *= 2)
//            {
//                offset >>= 1;

//                __syncthreads();
//                if (x < d)
//                {
//                    int ai = offset * 2 * x + 1 * offset - 1;
//                    int bi = offset * 2 * x + 2 * offset - 1;

//                    W t = line[ai];
//                    line[ai] = line[bi];
//                    line[bi] += t;
//                }
//            }
//            __syncthreads();

//            // calculate sum
//            int start = 0;
//            int out_idx = 0;
//            int end = start + fx;
//            while (start < (blockDim.x << 1) && end < (blockDim.x << 1))
//            {
//                buffer(y, out_idx) = saturate_cast<T>((line[end] - line[start]) / fx);
//                start = end;
//                end = start + fx;
//                out_idx++;
//            }

//        }

        template <typename T>
        __device__ void scan_y(DevMem2D_<typename scan_traits<T>::scan_line_type> buffer,int fx, int fy,  DevMem2D_<T> dst,
                               typename scan_traits<T>::scan_line_type* line, int g_base)
        {
            typedef typename scan_traits<T>::scan_line_type W;

            const int y = threadIdx.x;
            const int x = blockIdx.x;

            float scale = 1.f / (fx * fy);

            if (x >= buffer.cols) return;

            int offset = 1;
            line[2 * y + 0] = buffer((g_base * fy) + 2 * y + 1, x);

            if (y != (blockDim.x -1) )
                line[2 * y + 1] = buffer((g_base * fy) + 2 * y + 2, x);
            else
                line[2 * y + 1] = 0;

            __syncthreads();

            // reduction
            for (int d = blockDim.x; d > 0; d >>= 1)
            {
                __syncthreads();
                if (y < d)
                {
                    int ai = 2 * y * offset -1 + 1 * offset;
                    int bi = 2 * y * offset -1 + 2 * offset;
                    line[bi] += line[ai];
                }

                offset *= 2;
            }

            __syncthreads();
            // convolution
            if (y == 0) line[(blockDim.x << 1) - 1] = (W)buffer(0, x);

            for (int d = 1; d < (blockDim.x << 1); d *= 2)
            {
                offset >>= 1;

                __syncthreads();
                if (y < d)
                {
                    int ai = offset * 2 * y + 1 * offset - 1;
                    int bi = offset * 2 * y + 2 * offset - 1;


                    W t = line[ai];
                    line[ai] = line[bi];
                    line[bi] += t;
                }
            }
            __syncthreads();

            if (y < dst.rows)
            {
                W start = (y == 0)? (W)0:line[y * fy -1];
                W end = line[y * fy + fy - 1];
                dst(g_base +  y ,x) = saturate_cast<T>((end - start) * scale);
            }
        }

        template <typename T>
        __device__ void scan_x(const DevMem2D_<T> src, int fx, int fy, DevMem2D_<typename scan_traits<T>::scan_line_type> buffer,
                               typename scan_traits<T>::scan_line_type* line, int g_base)
        {
            typedef typename scan_traits<T>::scan_line_type W;

            const int x = threadIdx.x;
            const int y = blockIdx.x;

            float scale = 1.f / (fx * fy);

            if (y >= src.rows) return;

            int offset = 1;

            line[2 * x + 0] = (W)src(y, (g_base * fx) + 2 * x + 1);

            if (x != (blockDim.x -1) )
                line[2 * x + 1] = (W)src(y, (g_base * fx) + 2 * x + 2);
            else
                line[2 * x + 1] = 0;

            __syncthreads();

            // reduction
            for (int d = blockDim.x; d > 0; d >>= 1)
            {
                __syncthreads();
                if (x < d)
                {
                    int ai = 2 * x * offset -1 + 1 * offset;
                    int bi = 2 * x * offset -1 + 2 * offset;
                    line[bi] += line[ai];
                }

                offset *= 2;
            }

            __syncthreads();
            // convolution
            if (x == 0) line[(blockDim.x << 1) - 1] = (W)src(y, 0);

            for (int d = 1; d < (blockDim.x << 1); d *= 2)
            {
                offset >>= 1;

                __syncthreads();
                if (x < d)
                {
                    int ai = offset * 2 * x + 1 * offset - 1;
                    int bi = offset * 2 * x + 2 * offset - 1;

                    W t = line[ai];
                    line[ai] = line[bi];
                    line[bi] += t;
                }
            }
            __syncthreads();

            if (x < buffer.cols)
            {
                W start = (x == 0)? (W)0:line[x * fx -1];
                W end = line[x * fx + fx - 1];
                buffer(y, g_base +  x) =(end - start);
            }
        }

        template <typename T>
        __global__ void resize_area_scan_x(const DevMem2D_<T> src, DevMem2D_<T> dst, int fx, int fy,  DevMem2D_<typename scan_traits<T>::scan_line_type> buffer)
        {
            typedef typename scan_traits<T>::scan_line_type W;
            extern __shared__ W line[];
            scan_x(src,fx,fy, buffer,line, 0);
        }

        template <typename T>
        __global__ void resize_area_scan_y(const DevMem2D_<T> src, DevMem2D_<T> dst, int fx, int fy,  DevMem2D_<typename scan_traits<T>::scan_line_type> buffer)
        {
            typedef typename scan_traits<T>::scan_line_type W;
            extern __shared__ W line[];
            scan_y(buffer,fx, fy, dst, line, 0);
        }

        template <typename T> struct InterAreaDispatcherStream
        {
            static void call(const DevMem2D_<T> src, int fx, int fy, DevMem2D_<T> dst, DevMem2D_<typename scan_traits<T>::scan_line_type> buffer, hipStream_t stream)
            {
                resize_area_scan_x<T><<<src.rows, (src.cols >> 1), src.cols * sizeof(typename scan_traits<T>::scan_line_type) >>>(src, dst, fx, fy, buffer);

                resize_area_scan_y<T><<<dst.cols, (src.rows >> 1), src.rows * sizeof(typename scan_traits<T>::scan_line_type) >>>(src, dst, fx, fy, buffer);
                cudaSafeCall( hipGetLastError() );

                if (stream == 0)
                    cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        template <typename T>
        void resize_area_gpu(const DevMem2Db src, DevMem2Db dst,float fx, float fy,
                             int interpolation, DevMem2Df buffer, hipStream_t stream)
        {
            (void)interpolation;

            int iscale_x = round(fx);
            int iscale_y = round(fy);

            InterAreaDispatcherStream<T>::call(src, iscale_x, iscale_y, dst, buffer, stream);
        }

        template void resize_area_gpu<uchar>(DevMem2Db src, DevMem2Db dst, float fx, float fy, int interpolation, DevMem2Df buffer, hipStream_t stream);

    } // namespace imgproc
}}} // namespace cv { namespace gpu { namespace device
