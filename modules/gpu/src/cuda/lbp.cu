#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <opencv2/gpu/device/lbp.hpp>
#include <opencv2/gpu/device/vec_traits.hpp>
#include <opencv2/gpu/device/saturate_cast.hpp>

namespace cv { namespace gpu { namespace device
{
    namespace lbp
    {

        texture<int, hipTextureType2D, hipReadModeElementType> tintegral(false, hipFilterModePoint, hipAddressModeClamp);

        struct LBP
        {
            __device__ __forceinline__ LBP(const LBP& other) {(void)other;}
            __device__ __forceinline__ LBP() {}

            //feature as uchar x, y - left top, z,w - right bottom
            __device__ __forceinline__ int operator() (int ty, int tx, int fh, int featurez, int& shift) const
            {
                int anchors[9];

                anchors[0]  = tex2D(tintegral, tx, ty);
                anchors[1]  = tex2D(tintegral, tx + featurez, ty);
                anchors[0] -= anchors[1];
                anchors[2]  = tex2D(tintegral, tx + featurez * 2, ty);
                anchors[1] -= anchors[2];
                anchors[2] -= tex2D(tintegral, tx + featurez * 3, ty);

                ty += fh;
                anchors[3]  = tex2D(tintegral, tx, ty);
                anchors[4]  = tex2D(tintegral, tx + featurez, ty);
                anchors[3] -= anchors[4];
                anchors[5]  = tex2D(tintegral, tx + featurez * 2, ty);
                anchors[4] -= anchors[5];
                anchors[5] -= tex2D(tintegral, tx + featurez * 3, ty);

                anchors[0] -= anchors[3];
                anchors[1] -= anchors[4];
                anchors[2] -= anchors[5];
                // 0 - 2 contains s0 - s2

                ty += fh;
                anchors[6]  = tex2D(tintegral, tx, ty);
                anchors[7]  = tex2D(tintegral, tx + featurez, ty);
                anchors[6] -= anchors[7];
                anchors[8]  = tex2D(tintegral, tx + featurez * 2, ty);
                anchors[7] -= anchors[8];
                anchors[8] -= tex2D(tintegral, tx + featurez * 3, ty);

                anchors[3] -= anchors[6];
                anchors[4] -= anchors[7];
                anchors[5] -= anchors[8];
                // 3 - 5 contains s3 - s5

                anchors[0] -= anchors[4];
                anchors[1] -= anchors[4];
                anchors[2] -= anchors[4];
                anchors[3] -= anchors[4];
                anchors[5] -= anchors[4];

                int response = (~(anchors[0] >> 31)) & 4;
                response |= (~(anchors[1] >> 31)) & 2;;
                response |= (~(anchors[2] >> 31)) & 1;

                shift = (~(anchors[5] >> 31)) & 16;
                shift |= (~(anchors[3] >> 31)) & 1;

                ty += fh;
                anchors[0]  = tex2D(tintegral, tx, ty);
                anchors[1]  = tex2D(tintegral, tx + featurez, ty);
                anchors[0] -= anchors[1];
                anchors[2]  = tex2D(tintegral, tx + featurez * 2, ty);
                anchors[1] -= anchors[2];
                anchors[2] -= tex2D(tintegral, tx + featurez * 3, ty);

                anchors[6] -= anchors[0];
                anchors[7] -= anchors[1];
                anchors[8] -= anchors[2];
                // 0 -2 contains s6 - s8

                anchors[6] -= anchors[4];
                anchors[7] -= anchors[4];
                anchors[8] -= anchors[4];

                shift |= (~(anchors[6] >> 31)) & 2;
                shift |= (~(anchors[7] >> 31)) & 4;
                shift |= (~(anchors[8] >> 31)) & 8;
                return response;
            }
        };

        void bindIntegral(DevMem2Di integral)
        {
            hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
            cudaSafeCall( hipBindTexture2D(0, &tintegral, integral.ptr(), &desc, (size_t)integral.cols, (size_t)integral.rows, (size_t)integral.step));
        }

        void unbindIntegral()
        {
             cudaSafeCall( hipUnbindTexture(&tintegral));
        }

        __global__ void lbp_classify_stump(const Stage* stages, const int nstages, const ClNode* nodes, const float* leaves, const int* subsets, const uchar4* features,
           /* const int* integral,const int istep,  const int workWidth,const int workHeight,*/ const int clWidth, const int clHeight, const float scale, const int step,
            const int subsetSize, DevMem2D_<int4> objects, unsigned int* n)
        {
            int x = threadIdx.x * step;
            int y = blockIdx.x * step;

            int current_node = 0;
            int current_leave = 0;

            LBP evaluator;
            for (int s = 0; s < nstages; s++ )
            {
                float sum = 0;
                Stage stage = stages[s];
                for (int t = 0; t < stage.ntrees; t++)
                {
                    ClNode node = nodes[current_node];

                    uchar4 feature = features[node.featureIdx];
                    int shift;
                    int c = evaluator(y + feature.y, x + feature.x, feature.w, feature.z, shift);
                    int idx =  (subsets[ current_node * subsetSize + c] & ( 1 << shift)) ? current_leave : current_leave + 1;
                    sum += leaves[idx];
                    current_node += 1;
                    current_leave += 2;
                }
                if (sum < stage.threshold)
                    return;
            }

            int4 rect;
            rect.x = roundf(x * scale);
            rect.y = roundf(y * scale);
            rect.z = clWidth;
            rect.w = clHeight;
#if defined (__CUDA_ARCH__) && (__CUDA_ARCH__ < 120)
            int res = __atomicInc(n, 100U);
#else
            int res = atomicInc(n, 100U);
#endif
            objects(0, res) = rect;
        }

        template<typename Pr>
        __global__ void disjoin(int4* candidates, int4* objects, unsigned int n, int groupThreshold, float grouping_eps, unsigned int* nclasses)
        {
            using cv::gpu::device::VecTraits;
            unsigned int tid = threadIdx.x;
            extern __shared__ int sbuff[];

            int* labels = sbuff;
            int* rrects = (int*)(sbuff + n);

            Pr predicate(grouping_eps);
            partition(candidates, n, labels, predicate);

            rrects[tid * 4 + 0] = 0;
            rrects[tid * 4 + 1] = 0;
            rrects[tid * 4 + 2] = 0;
            rrects[tid * 4 + 3] = 0;
            __syncthreads();

            int cls = labels[tid];
#if defined (__CUDA_ARCH__) && (__CUDA_ARCH__ < 120)
            __atomicAdd((int*)(rrects + cls * 4 + 0), candidates[tid].x);
            __atomicAdd((int*)(rrects + cls * 4 + 1), candidates[tid].y);
            __atomicAdd((int*)(rrects + cls * 4 + 2), candidates[tid].z);
            __atomicAdd((int*)(rrects + cls * 4 + 3), candidates[tid].w);
#else
            atomicAdd((int*)(rrects + cls * 4 + 0), candidates[tid].x);
            atomicAdd((int*)(rrects + cls * 4 + 1), candidates[tid].y);
            atomicAdd((int*)(rrects + cls * 4 + 2), candidates[tid].z);
            atomicAdd((int*)(rrects + cls * 4 + 3), candidates[tid].w);
#endif
            labels[tid] = 0;
            __syncthreads();
#if defined (__CUDA_ARCH__) && (__CUDA_ARCH__ < 120)
            __atomicInc((unsigned int*)labels + cls, n);
#else
            atomicInc((unsigned int*)labels + cls, n);
#endif
            *nclasses = 0;

            int active = labels[tid];
            if (active)
            {
                int* r1 = rrects + tid * 4;
                float s = 1.f / active;
                r1[0] = saturate_cast<int>(r1[0] * s);
                r1[1] = saturate_cast<int>(r1[1] * s);
                r1[2] = saturate_cast<int>(r1[2] * s);
                r1[3] = saturate_cast<int>(r1[3] * s);

                int n1 = active;
                __syncthreads();
                unsigned int j = 0;
                if( active > groupThreshold )
                {
                    for (j = 0; j < n; j++)
                    {
                        int n2 = labels[j];
                        if(!n2 || j == tid || n2 <= groupThreshold )
                        continue;

                        int* r2 = rrects + j * 4;

                        int dx = saturate_cast<int>( r2[2] * grouping_eps );
                        int dy = saturate_cast<int>( r2[3] * grouping_eps );

                        if( tid != j && r1[0] >= r2[0] - dx && r1[1] >= r2[1] - dy &&
                            r1[0] + r1[2] <= r2[0] + r2[2] + dx && r1[1] + r1[3] <= r2[1] + r2[3] + dy &&
                            (n2 > max(3, n1) || n1 < 3) )
                            break;
                    }
                    if( j == n)
                    {
#if defined (__CUDA_ARCH__) && (__CUDA_ARCH__ < 120)
                        objects[__atomicInc(nclasses, n)] = VecTraits<int4>::make(r1[0], r1[1], r1[2], r1[3]);
#else
                        objects[atomicInc(nclasses, n)] = VecTraits<int4>::make(r1[0], r1[1], r1[2], r1[3]);
#endif
                    }
                }
            }
        }

        void classifyStump(const DevMem2Db& mstages, const int nstages, const DevMem2Di& mnodes, const DevMem2Df& mleaves, const DevMem2Di& msubsets, const DevMem2Db& mfeatures,
                           /*const DevMem2Di& integral,*/ const int workWidth, const int workHeight, const int clWidth, const int clHeight, float scale, int step, int subsetSize,
                           DevMem2D_<int4> objects, unsigned int* classified)
        {
            int blocks  = ceilf(workHeight / (float)step);
            int threads = ceilf(workWidth / (float)step);

            Stage* stages = (Stage*)(mstages.ptr());
            ClNode* nodes = (ClNode*)(mnodes.ptr());
            const float* leaves = mleaves.ptr();
            const int* subsets = msubsets.ptr();
            const uchar4* features = (uchar4*)(mfeatures.ptr());
            lbp_classify_stump<<<blocks, threads>>>(stages, nstages, nodes, leaves, subsets, features, /*integ, istep,
                workWidth, workHeight,*/ clWidth, clHeight, scale, step, subsetSize, objects, classified);
        }

        int connectedConmonents(DevMem2D_<int4> candidates, DevMem2D_<int4> objects, int groupThreshold, float grouping_eps, unsigned int* nclasses)
        {
            int threads = candidates.cols;
            int smem_amount = threads * sizeof(int) + threads * sizeof(int4);
            disjoin<InSameComponint><<<1, threads, smem_amount>>>((int4*)candidates.ptr(), (int4*)objects.ptr(), candidates.cols, groupThreshold, grouping_eps, nclasses);
            return 0;
        }
    }
}}}