#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/filters.hpp"
# include <cfloat>

namespace cv { namespace gpu { namespace device
{
    namespace imgproc
    {
        template <typename Ptr2D, typename T> __global__ void resize(const Ptr2D src, float fx, float fy, DevMem2D_<T> dst)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                const float xcoo = x * fx;
                const float ycoo = y * fy;

                dst(y, x) = saturate_cast<T>(src(ycoo, xcoo));
            }
        }

        template <typename Ptr2D, typename T> __global__ void resize_area(const Ptr2D src, float fx, float fy, DevMem2D_<T> dst)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                dst(y, x) = saturate_cast<T>(src(y, x));
            }
        }

        template <template <typename> class Filter, typename T> struct ResizeDispatcherStream
        {
            static void call(DevMem2D_<T> src, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream)
            {
                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                BrdReplicate<T> brd(src.rows, src.cols);
                BorderReader< PtrStep<T>, BrdReplicate<T> > brdSrc(src, brd);
                Filter< BorderReader< PtrStep<T>, BrdReplicate<T> > > filteredSrc(brdSrc, fx, fy);

                resize<<<grid, block, 0, stream>>>(filteredSrc, fx, fy, dst);
                cudaSafeCall( hipGetLastError() );
            }
        };

        template <typename T> struct ResizeDispatcherStream<AreaFilter, T>
        {
            static void call(DevMem2D_<T> src, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream)
            {
                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                BrdConstant<T> brd(src.rows, src.cols);
                BorderReader< PtrStep<T>, BrdConstant<T> > brdSrc(src, brd);
                AreaFilter< BorderReader< PtrStep<T>, BrdConstant<T> > > filteredSrc(brdSrc, fx, fy);
                resize_area<<<grid, block, 0, stream>>>(filteredSrc, fx, fy, dst);
                cudaSafeCall( hipGetLastError() );
                if (stream == 0)
                    cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        template <typename T> struct ResizeDispatcherStream<IntegerAreaFilter, T>
        {
            static void call(DevMem2D_<T> src, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream)
            {
                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));
                BrdConstant<T> brd(src.rows, src.cols);
                BorderReader< PtrStep<T>, BrdConstant<T> > brdSrc(src, brd);
                IntegerAreaFilter< BorderReader< PtrStep<T>, BrdConstant<T> > > filteredSrc(brdSrc, fx, fy);
                resize_area<<<grid, block, 0, stream>>>(filteredSrc, fx, fy, dst);
                cudaSafeCall( hipGetLastError() );
                if (stream == 0)
                    cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        template <template <typename> class Filter, typename T> struct ResizeDispatcherNonStream
        {
            static void call(DevMem2D_<T> src, DevMem2D_<T> srcWhole, int xoff, int yoff, float fx, float fy, DevMem2D_<T> dst)
            {
                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                BrdReplicate<T> brd(src.rows, src.cols);
                BorderReader< PtrStep<T>, BrdReplicate<T> > brdSrc(src, brd);
                Filter< BorderReader< PtrStep<T>, BrdReplicate<T> > > filteredSrc(brdSrc);

                resize<<<grid, block>>>(filteredSrc, fx, fy, dst);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        #define OPENCV_GPU_IMPLEMENT_RESIZE_TEX(type) \
            texture< type , hipTextureType2D> tex_resize_ ## type (0, hipFilterModePoint, hipAddressModeClamp); \
            struct tex_resize_ ## type ## _reader \
            { \
                typedef type elem_type; \
                typedef int index_type; \
                const int xoff; \
                const int yoff; \
                __host__ tex_resize_ ## type ## _reader(int xoff_, int yoff_) : xoff(xoff_), yoff(yoff_) {} \
                __device__ __forceinline__ elem_type operator ()(index_type y, index_type x) const \
                { \
                    return tex2D(tex_resize_ ## type, x + xoff, y + yoff); \
                } \
            }; \
            template <template <typename> class Filter> struct ResizeDispatcherNonStream<Filter, type > \
            { \
                static void call(DevMem2D_< type > src, DevMem2D_< type > srcWhole, int xoff, int yoff, float fx, float fy, DevMem2D_< type > dst) \
                { \
                    dim3 block(32, 8); \
                    dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y)); \
                    bindTexture(&tex_resize_ ## type, srcWhole); \
                    tex_resize_ ## type ## _reader texSrc(xoff, yoff); \
                    if (srcWhole.cols == src.cols && srcWhole.rows == src.rows) \
                    { \
                        Filter<tex_resize_ ## type ## _reader> filteredSrc(texSrc); \
                        resize<<<grid, block>>>(filteredSrc, fx, fy, dst); \
                    } \
                    else \
                    { \
                        BrdReplicate< type > brd(src.rows, src.cols); \
                        BorderReader<tex_resize_ ## type ## _reader, BrdReplicate< type > > brdSrc(texSrc, brd); \
                        Filter< BorderReader<tex_resize_ ## type ## _reader, BrdReplicate< type > > > filteredSrc(brdSrc); \
                        resize<<<grid, block>>>(filteredSrc, fx, fy, dst); \
                    } \
                    cudaSafeCall( hipGetLastError() ); \
                    cudaSafeCall( hipDeviceSynchronize() ); \
                } \
            };

        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(uchar)
        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(uchar4)

        //OPENCV_GPU_IMPLEMENT_RESIZE_TEX(schar)
        //OPENCV_GPU_IMPLEMENT_RESIZE_TEX(char4)

        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(ushort)
        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(ushort4)

        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(short)
        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(short4)

        //OPENCV_GPU_IMPLEMENT_RESIZE_TEX(int)
        //OPENCV_GPU_IMPLEMENT_RESIZE_TEX(int4)

        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(float)
        OPENCV_GPU_IMPLEMENT_RESIZE_TEX(float4)

        #undef OPENCV_GPU_IMPLEMENT_RESIZE_TEX

        template <template <typename> class Filter, typename T> struct ResizeDispatcher
        {
            static void call(DevMem2D_<T> src, DevMem2D_<T> srcWhole, int xoff, int yoff, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream)
            {
                if (stream == 0)
                    ResizeDispatcherNonStream<Filter, T>::call(src, srcWhole, xoff, yoff, fx, fy, dst);
                else
                    ResizeDispatcherStream<Filter, T>::call(src, fx, fy, dst, stream);
            }
        };

        template <typename T> struct ResizeDispatcher<AreaFilter, T>
        {
            static void call(DevMem2D_<T> src, DevMem2D_<T> srcWhole, int xoff, int yoff, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream)
            {
                int iscale_x = round(fx);
                int iscale_y = round(fy);

                if( std::abs(fx - iscale_x) < FLT_MIN && std::abs(fy - iscale_y) < FLT_MIN)
                    ResizeDispatcherStream<IntegerAreaFilter, T>::call(src, fx, fy, dst, stream);
                else
                    ResizeDispatcherStream<AreaFilter, T>::call(src, fx, fy, dst, stream);
            }
        };

        template <typename T> void resize_gpu(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy,
            DevMem2Db dst, int interpolation, hipStream_t stream)
        {
            typedef void (*caller_t)(DevMem2D_<T> src, DevMem2D_<T> srcWhole, int xoff, int yoff, float fx, float fy, DevMem2D_<T> dst, hipStream_t stream);

            static const caller_t callers[4] =
            {
                ResizeDispatcher<PointFilter, T>::call,
                ResizeDispatcher<LinearFilter, T>::call,
                ResizeDispatcher<CubicFilter, T>::call,
                ResizeDispatcher<AreaFilter, T>::call
            };
            // chenge to linear if area interpolation upscaling
            if (interpolation == 3 && (fx <= 1.f || fy <= 1.f))
                interpolation = 1;

            callers[interpolation](static_cast< DevMem2D_<T> >(src), static_cast< DevMem2D_<T> >(srcWhole), xoff, yoff, fx, fy,
                static_cast< DevMem2D_<T> >(dst), stream);
        }

        template void resize_gpu<uchar >(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<uchar2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<uchar3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<uchar4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        //template void resize_gpu<schar>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<char2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<char3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<char4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        template void resize_gpu<ushort >(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<ushort2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<ushort3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<ushort4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        template void resize_gpu<short >(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<short2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<short3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<short4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        //template void resize_gpu<int >(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<int2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<int3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<int4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        template void resize_gpu<float >(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        //template void resize_gpu<float2>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<float3>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);
        template void resize_gpu<float4>(DevMem2Db src, DevMem2Db srcWhole, int xoff, int yoff, float fx, float fy, DevMem2Db dst, int interpolation, hipStream_t stream);

        template<typename T> struct scan_traits{};

        template<> struct scan_traits<uchar>
        {
            typedef float scan_line_type;
        };

//        template <typename T>
//        __global__ void resize_area_scan(const DevMem2D_<T> src, DevMem2D_<T> dst, int fx, int fy,  DevMem2D_<T> buffer)
//        {
//            typedef typename scan_traits<T>::scan_line_type W;
//            extern __shared__ W line[];

//            const int x = threadIdx.x;
//            const int y = blockIdx.x;

//            if (y >= src.rows) return;

//            int offset = 1;

//            line[2 * x + 0] = src(y, 2 * x + 0);
//            line[2 * x + 1] = src(y, 2 * x + 1);

//            __syncthreads();//???
//            // reduction
//            for (int d = blockDim.x; d > 0; d >>= 1)
//            {
//                __syncthreads();
//                if (x < d)
//                {
//                    int ai = 2 * x * offset -1 + 1 * offset;
//                    int bi = 2 * x * offset -1 + 2 * offset;
//                    line[bi] += line[ai];
//                }

//                offset *= 2;
//            }

//            __syncthreads();
//            // convolution
//            if (x == 0) { line[(blockDim.x << 1) - 1] = 0; printf("offset: %d!!!!!!!!!!!!!\n", fx);}

//            for (int d = 1; d < (blockDim.x << 1); d *= 2)
//            {
//                offset >>= 1;

//                __syncthreads();
//                if (x < d)
//                {
//                    int ai = offset * 2 * x + 1 * offset - 1;
//                    int bi = offset * 2 * x + 2 * offset - 1;

//                    W t = line[ai];
//                    line[ai] = line[bi];
//                    line[bi] += t;
//                }
//            }
//            __syncthreads();

//            // calculate sum
//            int start = 0;
//            int out_idx = 0;
//            int end = start + fx;
//            while (start < (blockDim.x << 1) && end < (blockDim.x << 1))
//            {
//                buffer(y, out_idx) = saturate_cast<T>((line[end] - line[start]) / fx);
//                start = end;
//                end = start + fx;
//                out_idx++;
//            }

//        }

        template <typename T>
        __device__ void scan_y(DevMem2D_<typename scan_traits<T>::scan_line_type> buffer,int fx, int fy,  DevMem2D_<T> dst,
                               typename scan_traits<T>::scan_line_type* line, int g_base)
        {
            typedef typename scan_traits<T>::scan_line_type W;

            const int y = threadIdx.x;
            const int x = blockIdx.x;

            float scale = 1.f / (fx * fy);

            if (x >= buffer.cols) return;

            int offset = 1;
            line[2 * y + 0] = buffer((g_base * fy) + 2 * y + 1, x);

            if (y != (blockDim.x -1) )
                line[2 * y + 1] = buffer((g_base * fy) + 2 * y + 2, x);
            else
                line[2 * y + 1] = 0;

            __syncthreads();

            // reduction
            for (int d = blockDim.x; d > 0; d >>= 1)
            {
                __syncthreads();
                if (y < d)
                {
                    int ai = 2 * y * offset -1 + 1 * offset;
                    int bi = 2 * y * offset -1 + 2 * offset;
                    line[bi] += line[ai];
                }

                offset *= 2;
            }

            __syncthreads();
            // convolution
            if (y == 0) line[(blockDim.x << 1) - 1] = (W)buffer(0, x);

            for (int d = 1; d < (blockDim.x << 1); d *= 2)
            {
                offset >>= 1;

                __syncthreads();
                if (y < d)
                {
                    int ai = offset * 2 * y + 1 * offset - 1;
                    int bi = offset * 2 * y + 2 * offset - 1;


                    W t = line[ai];
                    line[ai] = line[bi];
                    line[bi] += t;
                }
            }
            __syncthreads();

            if (y < dst.rows)
            {
                W start = (y == 0)? (W)0:line[y * fy -1];
                W end = line[y * fy + fy - 1];
                dst(g_base +  y ,x) = saturate_cast<T>((end - start) * scale);
            }
        }

        template <typename T>
        __device__ void scan_x(const DevMem2D_<T> src, int fx, int fy, DevMem2D_<typename scan_traits<T>::scan_line_type> buffer,
                               typename scan_traits<T>::scan_line_type* line, int g_base)
        {
            typedef typename scan_traits<T>::scan_line_type W;

            const int x = threadIdx.x;
            const int y = blockIdx.x;

            float scale = 1.f / (fx * fy);

            if (y >= src.rows) return;

            int offset = 1;

            line[2 * x + 0] = (W)src(y, (g_base * fx) + 2 * x + 1);

            if (x != (blockDim.x -1) )
                line[2 * x + 1] = (W)src(y, (g_base * fx) + 2 * x + 2);
            else
                line[2 * x + 1] = 0;

            __syncthreads();

            // reduction
            for (int d = blockDim.x; d > 0; d >>= 1)
            {
                __syncthreads();
                if (x < d)
                {
                    int ai = 2 * x * offset -1 + 1 * offset;
                    int bi = 2 * x * offset -1 + 2 * offset;
                    line[bi] += line[ai];
                }

                offset *= 2;
            }

            __syncthreads();
            // convolution
            if (x == 0) line[(blockDim.x << 1) - 1] = (W)src(y, 0);

            for (int d = 1; d < (blockDim.x << 1); d *= 2)
            {
                offset >>= 1;

                __syncthreads();
                if (x < d)
                {
                    int ai = offset * 2 * x + 1 * offset - 1;
                    int bi = offset * 2 * x + 2 * offset - 1;

                    W t = line[ai];
                    line[ai] = line[bi];
                    line[bi] += t;
                }
            }
            __syncthreads();

            if (x < buffer.cols)
            {
                W start = (x == 0)? (W)0:line[x * fx -1];
                W end = line[x * fx + fx - 1];
                buffer(y, g_base +  x) =(end - start);
            }
        }

        enum ScanKind { exclusive,  inclusive } ;

        template <ScanKind Kind , class T>
        __device__ __forceinline__ T scan_warp ( volatile T *ptr , const unsigned int idx = threadIdx.x )
        {
            const unsigned int lane = idx & 31;

            if ( lane >=  1) ptr [idx ] = ptr [idx -  1] + ptr [idx];
            if ( lane >=  2) ptr [idx ] = ptr [idx -  2] + ptr [idx];
            if ( lane >=  4) ptr [idx ] = ptr [idx -  4] + ptr [idx];
            if ( lane >=  8) ptr [idx ] = ptr [idx -  8] + ptr [idx];
            if ( lane >= 16) ptr [idx ] = ptr [idx - 16] + ptr [idx];

            if( Kind == inclusive )
                return ptr [idx ];
            else
                return (lane > 0) ? ptr [idx - 1] : 0;
        }

        template <ScanKind Kind , class T>
        __device__ __forceinline__ T scan_block( volatile T *ptr)
        {
            const unsigned int idx = threadIdx.x;
            const unsigned int lane = idx & 31;
            const unsigned int warp = idx >> 5;

            T val = scan_warp <Kind>( ptr , idx );
            __syncthreads ();

            if( lane == 31 )
                ptr [ warp ] = ptr [idx ];

            __syncthreads ();

            if( warp == 0 )
                scan_warp<inclusive>( ptr , idx );

            __syncthreads ();

            if ( warp > 0)
                val = ptr [warp -1] + val;

            __syncthreads ();

            ptr[idx] = val;

            __syncthreads ();

            return val ;
        }

        template<typename T, typename W>
        __global__ void resise_scan_fast_x(const DevMem2D_<T> src, DevMem2D_<W> dst, int fx, int fy, int thred_lines, int stride)
        {
            extern __shared__ W sbuf[];

            const unsigned int tid = threadIdx. x;

            // load line-block on shared memory
            int y = blockIdx.x / thred_lines;
            int input_stride = (blockIdx.x % thred_lines) * stride;
            int x = input_stride  + tid;

            // store global data in shared memory
            if (x  < src.cols && y < src.rows)
                sbuf[tid] = src(y, x);
            else
                sbuf[tid] = 0;
            __syncthreads();

            scan_block<inclusive, W>(sbuf);

            float scale =  __fdividef(1.f, fx);
            int out_stride = input_stride / fx;
            int count = blockDim.x / fx;

            if (tid < count)
            {
                int start_idx = (tid == 0)? 0 : tid * fx - 1;
                int end_idx = tid * fx + fx - 1;

                W start = (tid == 0)? (W)0:sbuf[start_idx];
                W end = sbuf[end_idx];

                if (blockIdx.x == 0)
                    printf("%d~~~~~~~~ start_idx %d, end_idx %d, start %f, end %f\n",
                           tid, start_idx, end_idx, start, end);

                dst(y, out_stride  +  tid) = (end - start);
            }
        }

        template<typename T, typename W>
        __global__ void resise_scan_fast_y(const DevMem2D_<W> src, DevMem2D_<T> dst, int fx, int fy, int thred_lines, int stride)
        {
            extern __shared__ W sbuf[];

            const unsigned int tid = threadIdx. x;

            // load line-block on shared memory
            int x = blockIdx.x / thred_lines;

            int global_stride = (blockIdx.x % thred_lines) * stride;
            int y = global_stride + tid;

            // store global data in shared memory
            if (x  < src.cols && y < src.rows)
                sbuf[tid] = src(y, x);
            else
                sbuf[tid] = 0;

            __syncthreads();
            scan_block<inclusive, W>(sbuf);

            float scale =  __fdividef(1.f, fx * fy);
            int out_stride = global_stride / fx;
            int count = blockDim.x / fx;

            if (tid < count)
            {
                int start_idx = (tid == 0)? 0 : tid * fx - 1;
                int end_idx = tid * fx + fx - 1;

                W start = (tid == 0)? (W)0:sbuf[start_idx];
                W end = sbuf[end_idx];

                if (blockIdx.x == 0)
                    printf("!!!!!!!!%d~~~~~~~~ start_idx %d, end_idx %d, start %f, end %f\n",
                           tid, start_idx, end_idx, start, end);

                dst(out_stride  +  tid, x) = saturate_cast<T>((end - start) * scale);
            }
        }

        template <typename T>
        void resize_area_gpu(const DevMem2Db src, DevMem2Db dst,float fx, float fy,
                             int interpolation, DevMem2Df buffer, hipStream_t stream)
        {
            (void)interpolation;

            int iscale_x = round(fx);
            int iscale_y = round(fy);

            int warps = 4;
            const int threads = 32 * warps;
            int input_stride = threads / iscale_x;

            int thred_lines = divUp(src.cols, input_stride * iscale_x);
            int blocks = src.rows * thred_lines;

            printf("device code executed for X coordinate with:\nsize %d warps %d, threads %d, thred_lines %d, blocks %d input strude %d\n",
                   src.cols, warps, threads, thred_lines, blocks, input_stride * iscale_x);

            typedef typename scan_traits<T>::scan_line_type smem_type;

            resise_scan_fast_x<T, smem_type><<<blocks, threads, warps * 32 * sizeof(smem_type)>>>
                    (src, buffer, iscale_x, iscale_y, thred_lines, input_stride * iscale_x);

            input_stride = threads / iscale_y;
            thred_lines = divUp(src.rows, input_stride * iscale_y);
            blocks = dst.cols * thred_lines;

            printf("device code executed for Y coordinate with:\nsize %d warps %d, threads %d, thred_lines %d, blocks %d\n",
                   dst.rows, warps, threads, thred_lines, blocks);

            resise_scan_fast_y<T, smem_type><<<blocks, threads, warps * 32 * sizeof(smem_type)>>>
                    (buffer, dst, iscale_x, iscale_y, thred_lines, input_stride * iscale_y);

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void resize_area_gpu<uchar>(DevMem2Db src, DevMem2Db dst, float fx, float fy, int interpolation, DevMem2Df buffer, hipStream_t stream);

    } // namespace imgproc
}}} // namespace cv { namespace gpu { namespace device
