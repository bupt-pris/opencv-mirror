#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/gpu/device/functional.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/transform.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "internal_shared.hpp"

namespace cv { namespace gpu { namespace device
{
    //////////////////////////////////////////////////////////////////////////////////////
    // Compare

    template <typename T> struct NotEqual : binary_function<T, T, uchar>
    {
        __device__ __forceinline__ uchar operator()(T src1, T src2) const
        {
            return static_cast<uchar>(static_cast<int>(src1 != src2) * 255);
        }
    };

    template <typename T>
    inline void compare_ne(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream)
    {
        NotEqual<T> op;
        transform(static_cast< DevMem2D_<T> >(src1), static_cast< DevMem2D_<T> >(src2), dst, op, stream);
    }

    void compare_ne_8uc4(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream)
    {
        compare_ne<uint>(src1, src2, dst, stream);
    }
    void compare_ne_32f(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream)
    {
        compare_ne<float>(src1, src2, dst, stream);
    }


    //////////////////////////////////////////////////////////////////////////
    // Unary bitwise logical matrix operations

    enum { UN_OP_NOT };

    template <typename T, int opid>
    struct UnOp;

    template <typename T>
    struct UnOp<T, UN_OP_NOT>
    { 
        static __device__ __forceinline__ T call(T v) { return ~v; }
    };


    template <int opid>
    __global__ void bitwiseUnOpKernel(int rows, int width, const PtrStep src, PtrStep dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows) 
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src_ptr = src.ptr(y) + x;
            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = UnOp<uint, opid>::call(*(uint*)src_ptr);
            }
            else
            {
                const uchar* src_end = src.ptr(y) + width;
                while (src_ptr < src_end)
                {
                    *dst_ptr++ = UnOp<uchar, opid>::call(*src_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseUnOp(int rows, int width, const PtrStep src, PtrStep dst, 
                     hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)), 
                  divUp(rows, threads.y));

        bitwiseUnOpKernel<opid><<<grid, threads>>>(rows, width, src, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    template <typename T, int opid>
    __global__ void bitwiseUnOpKernel(int rows, int cols, int cn, const PtrStep src, 
                                      const PtrStep mask, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn]) 
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src_row = (const T*)src.ptr(y);

            dst_row[x] = UnOp<T, opid>::call(src_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseUnOp(int rows, int cols, int cn, const PtrStep src, 
                     const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseUnOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src, mask, dst); 
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    void bitwiseNotCaller(int rows, int cols, size_t elem_size1, int cn, 
                          const PtrStep src, PtrStep dst, hipStream_t stream)
    {
        bitwiseUnOp<UN_OP_NOT>(rows, static_cast<int>(cols * elem_size1 * cn), src, dst, stream);
    }


    template <typename T>
    void bitwiseMaskNotCaller(int rows, int cols, int cn, const PtrStep src, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseUnOp<T, UN_OP_NOT>(rows, cols * cn, cn, src, mask, dst, stream);
    }

    template void bitwiseMaskNotCaller<uchar>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskNotCaller<ushort>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskNotCaller<uint>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    //////////////////////////////////////////////////////////////////////////
    // Binary bitwise logical matrix operations

    enum { BIN_OP_OR, BIN_OP_AND, BIN_OP_XOR };

    template <typename T, int opid>
    struct BinOp;

    template <typename T>
    struct BinOp<T, BIN_OP_OR>
    { 
        static __device__ __forceinline__ T call(T a, T b) { return a | b; } 
    };


    template <typename T>
    struct BinOp<T, BIN_OP_AND>
    { 
        static __device__ __forceinline__ T call(T a, T b) { return a & b; } 
    };

    template <typename T>
    struct BinOp<T, BIN_OP_XOR>
    { 
        static __device__ __forceinline__ T call(T a, T b) { return a ^ b; } 
    };


    template <int opid>
    __global__ void bitwiseBinOpKernel(int rows, int width, const PtrStep src1, 
                                       const PtrStep src2, PtrStep dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows) 
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src1_ptr = src1.ptr(y) + x;
            const uchar* src2_ptr = src2.ptr(y) + x;

            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = BinOp<uint, opid>::call(*(uint*)src1_ptr, *(uint*)src2_ptr);
            }
            else
            {
                const uchar* src1_end = src1.ptr(y) + width;
                while (src1_ptr < src1_end)
                {
                    *dst_ptr++ = BinOp<uchar, opid>::call(*src1_ptr++, *src2_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseBinOp(int rows, int width, const PtrStep src1, const PtrStep src2, 
                      PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)), divUp(rows, threads.y));

        bitwiseBinOpKernel<opid><<<grid, threads>>>(rows, width, src1, src2, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    template <typename T, int opid>
    __global__ void bitwiseBinOpKernel(
            int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
            const PtrStep mask, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn]) 
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src1_row = (const T*)src1.ptr(y);
            const T* src2_row = (const T*)src2.ptr(y);

            dst_row[x] = BinOp<T, opid>::call(src1_row[x], src2_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseBinOp(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                        const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseBinOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src1, src2, mask, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    void bitwiseOrCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStep src1, 
                         const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_OR>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskOrCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                             const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_OR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskOrCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskOrCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskOrCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    void bitwiseAndCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStep src1, 
                          const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_AND>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskAndCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_AND>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskAndCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskAndCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskAndCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    void bitwiseXorCaller(int rows, int cols, size_t elem_size1, int cn, const PtrStep src1, 
                          const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_XOR>(rows, static_cast<int>(cols * elem_size1 * cn), src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskXorCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_XOR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskXorCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskXorCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskXorCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    //////////////////////////////////////////////////////////////////////////
    // min/max

    namespace detail
    {
        template <size_t size, typename F> struct MinMaxTraits : DefaultTransformFunctorTraits<F>
        {
        };
        template <typename F> struct MinMaxTraits<2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 4 };
        };
        template <typename F> struct MinMaxTraits<4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 4 };
            enum { smart_shift = 4 };
        };
    }

    template <typename T> struct TransformFunctorTraits< minimum<T> > : detail::MinMaxTraits< sizeof(T), minimum<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< maximum<T> > : detail::MinMaxTraits< sizeof(T), maximum<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< binder2nd< minimum<T> > > : detail::MinMaxTraits< sizeof(T), binder2nd< minimum<T> > >
    {
    };
    template <typename T> struct TransformFunctorTraits< binder2nd< maximum<T> > > : detail::MinMaxTraits< sizeof(T), binder2nd< maximum<T> > >
    {
    };
    
    template <typename T>
    void min_gpu(const DevMem2D_<T>& src1, const DevMem2D_<T>& src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        transform(src1, src2, dst, minimum<T>(), stream);    
    }

    template void min_gpu<uchar >(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream);
    template void min_gpu<schar >(const DevMem2D_<schar>& src1, const DevMem2D_<schar>& src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void min_gpu<ushort>(const DevMem2D_<ushort>& src1, const DevMem2D_<ushort>& src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void min_gpu<short >(const DevMem2D_<short>& src1, const DevMem2D_<short>& src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void min_gpu<int   >(const DevMem2D_<int>& src1, const DevMem2D_<int>& src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void min_gpu<float >(const DevMem2D_<float>& src1, const DevMem2D_<float>& src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void min_gpu<double>(const DevMem2D_<double>& src1, const DevMem2D_<double>& src2, const DevMem2D_<double>& dst, hipStream_t stream);

    template <typename T>
    void max_gpu(const DevMem2D_<T>& src1, const DevMem2D_<T>& src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        transform(src1, src2, dst, maximum<T>(), stream);    
    }
    
    template void max_gpu<uchar >(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream);
    template void max_gpu<schar >(const DevMem2D_<schar>& src1, const DevMem2D_<schar>& src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void max_gpu<ushort>(const DevMem2D_<ushort>& src1, const DevMem2D_<ushort>& src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void max_gpu<short >(const DevMem2D_<short>& src1, const DevMem2D_<short>& src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void max_gpu<int   >(const DevMem2D_<int>& src1, const DevMem2D_<int>& src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void max_gpu<float >(const DevMem2D_<float>& src1, const DevMem2D_<float>& src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void max_gpu<double>(const DevMem2D_<double>& src1, const DevMem2D_<double>& src2, const DevMem2D_<double>& dst, hipStream_t stream);

    template <typename T>
    void min_gpu(const DevMem2D_<T>& src1, T src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        transform(src1, dst, device::bind2nd(minimum<T>(), src2), stream);    
    }

    template void min_gpu<uchar >(const DevMem2D& src1, uchar src2, const DevMem2D& dst, hipStream_t stream);
    template void min_gpu<schar >(const DevMem2D_<schar>& src1, schar src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void min_gpu<ushort>(const DevMem2D_<ushort>& src1, ushort src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void min_gpu<short >(const DevMem2D_<short>& src1, short src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void min_gpu<int   >(const DevMem2D_<int>& src1, int src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void min_gpu<float >(const DevMem2D_<float>& src1, float src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void min_gpu<double>(const DevMem2D_<double>& src1, double src2, const DevMem2D_<double>& dst, hipStream_t stream);

    template <typename T>
    void max_gpu(const DevMem2D_<T>& src1, T src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        transform(src1, dst, device::bind2nd(maximum<T>(), src2), stream);    
    }

    template void max_gpu<uchar >(const DevMem2D& src1, uchar src2, const DevMem2D& dst, hipStream_t stream);
    template void max_gpu<schar >(const DevMem2D_<schar>& src1, schar src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void max_gpu<ushort>(const DevMem2D_<ushort>& src1, ushort src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void max_gpu<short >(const DevMem2D_<short>& src1, short src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void max_gpu<int   >(const DevMem2D_<int>& src1, int src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void max_gpu<float >(const DevMem2D_<float>& src1, float src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void max_gpu<double>(const DevMem2D_<double>& src1, double src2, const DevMem2D_<double>& dst, hipStream_t stream);

    
    //////////////////////////////////////////////////////////////////////////
    // threshold

    namespace detail
    {
        template <size_t size, typename F> struct ThresholdTraits : DefaultTransformFunctorTraits<F>
        {
        };
        template <typename F> struct ThresholdTraits<2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 4 };
        };
        template <typename F> struct ThresholdTraits<4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 4 };
            enum { smart_shift = 4 };
        };
    }

    template <typename T> struct TransformFunctorTraits< thresh_binary_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_binary_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_binary_inv_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_binary_inv_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_trunc_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_trunc_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_to_zero_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_to_zero_func<T> >
    {
    };
    template <typename T> struct TransformFunctorTraits< thresh_to_zero_inv_func<T> > : detail::ThresholdTraits< sizeof(T), thresh_to_zero_inv_func<T> >
    {
    };

    template <template <typename> class Op, typename T>
    void threshold_caller(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, T thresh, T maxVal, 
        hipStream_t stream)
    {
        Op<T> op(thresh, maxVal);
        transform(src, dst, op, stream);
    }

    template <typename T>
    void threshold_gpu(const DevMem2D& src, const DevMem2D& dst, T thresh, T maxVal, int type,
        hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, T thresh, T maxVal, 
            hipStream_t stream);

        static const caller_t callers[] = 
        {
            threshold_caller<thresh_binary_func, T>, 
            threshold_caller<thresh_binary_inv_func, T>, 
            threshold_caller<thresh_trunc_func, T>, 
            threshold_caller<thresh_to_zero_func, T>, 
            threshold_caller<thresh_to_zero_inv_func, T>
        };

        callers[type]((DevMem2D_<T>)src, (DevMem2D_<T>)dst, thresh, maxVal, stream);
    }

    template void threshold_gpu<uchar>(const DevMem2D& src, const DevMem2D& dst, uchar thresh, uchar maxVal, int type, hipStream_t stream);
    template void threshold_gpu<schar>(const DevMem2D& src, const DevMem2D& dst, schar thresh, schar maxVal, int type, hipStream_t stream);
    template void threshold_gpu<ushort>(const DevMem2D& src, const DevMem2D& dst, ushort thresh, ushort maxVal, int type, hipStream_t stream);
    template void threshold_gpu<short>(const DevMem2D& src, const DevMem2D& dst, short thresh, short maxVal, int type, hipStream_t stream);
    template void threshold_gpu<int>(const DevMem2D& src, const DevMem2D& dst, int thresh, int maxVal, int type, hipStream_t stream);
    template void threshold_gpu<float>(const DevMem2D& src, const DevMem2D& dst, float thresh, float maxVal, int type, hipStream_t stream);
    template void threshold_gpu<double>(const DevMem2D& src, const DevMem2D& dst, double thresh, double maxVal, int type, hipStream_t stream);


    //////////////////////////////////////////////////////////////////////////
    // subtract

    template <> struct TransformFunctorTraits< minus<short> > : DefaultTransformFunctorTraits< minus<short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T> void subtractCaller(const DevMem2D src1, const DevMem2D src2, DevMem2D dst, hipStream_t stream)
    {
        transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<T>)dst, minus<T>(), stream);
    }

    template void subtractCaller<short>(const DevMem2D src1, const DevMem2D src2, DevMem2D dst, hipStream_t stream);


    //////////////////////////////////////////////////////////////////////////
    // pow
    
    template<typename T, bool Signed = device::numeric_limits<T>::is_signed> struct PowOp : unary_function<T, T>
    {    
        float power;
        PowOp(float power_) : power(power_) {}
            
        __device__ __forceinline__ T operator()(const T& e) const
        {      
            return saturate_cast<T>(__powf((float)e, power));
        }      
    };

    template<typename T> struct PowOp<T, true> : unary_function<T, T>
    {
        float power;
        PowOp(float power_) : power(power_) {}

        __device__ __forceinline__ float operator()(const T& e) const
        {
            T res = saturate_cast<T>(__powf((float)e, power));            
            
            if ( (e < 0) && (1 & (int)power) )
                    res *= -1;            
            return res;         
        }
    };

    template<> struct PowOp<float> : unary_function<float, float>
    {
        float power;
        PowOp(float power_) : power(power_) {}

        __device__ __forceinline__ float operator()(const float& e) const
        {
            return __powf(::fabs(e), power);
        }
    };

    namespace detail
    {
        template <size_t size, typename T> struct PowOpTraits : DefaultTransformFunctorTraits< PowOp<T> >
        {
        };
        template <typename T> struct PowOpTraits<1, T> : DefaultTransformFunctorTraits< PowOp<T> >
        {
            enum { smart_block_dim_y = 8 };
            enum { smart_shift = 8 };
        };
        template <typename T> struct PowOpTraits<2, T> : DefaultTransformFunctorTraits< PowOp<T> >
        {
            enum { smart_shift = 4 };
        };
        template <typename T> struct PowOpTraits<4, T> : DefaultTransformFunctorTraits< PowOp<T> >
        {
            enum { smart_block_dim_y = 4 };
            enum { smart_shift = 4 };
        };
    }

    template <typename T> struct TransformFunctorTraits< PowOp<T> > : detail::PowOpTraits<sizeof(T), T>
    {
    };

    template<typename T>
    void pow_caller(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream)
    {
        transform((DevMem2D_<T>)src, (DevMem2D_<T>)dst, PowOp<T>(power), stream);
    }   

    template void pow_caller<uchar>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
    template void pow_caller<schar>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
    template void pow_caller<short>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
    template void pow_caller<ushort>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
    template void pow_caller<int>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);
    template void pow_caller<float>(const DevMem2D& src, float power, DevMem2D dst, hipStream_t stream);


    //////////////////////////////////////////////////////////////////////////
    // divide

    struct divide_8uc4_32f : binary_function<uchar4, float, uchar4>
    {
        __device__ __forceinline__ uchar4 operator ()(uchar4 a, float b) const
        {
            return make_uchar4(saturate_cast<uchar>(a.x / b), saturate_cast<uchar>(a.y / b),
                               saturate_cast<uchar>(a.z / b), saturate_cast<uchar>(a.w / b));
        }
    };

    template <> struct TransformFunctorTraits<divide_8uc4_32f> : DefaultTransformFunctorTraits<divide_8uc4_32f>
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void divide_gpu(const DevMem2D_<uchar4>& src1, const DevMem2Df& src2, const DevMem2D_<uchar4>& dst, hipStream_t stream)
    {
        transform(static_cast< DevMem2D_<uchar4> >(src1), src2, static_cast< DevMem2D_<uchar4> >(dst), divide_8uc4_32f(), stream);
    }


    struct divide_16sc4_32f : binary_function<short4, float, short4>
    {
        __device__ __forceinline__ short4 operator ()(short4 a, float b) const
        {
            return make_short4(saturate_cast<short>(a.x / b), saturate_cast<uchar>(a.y / b),
                               saturate_cast<short>(a.z / b), saturate_cast<uchar>(a.w / b));
        }
    };

    template <> struct TransformFunctorTraits<divide_16sc4_32f> : DefaultTransformFunctorTraits<divide_16sc4_32f>
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void divide_gpu(const DevMem2D_<short4>& src1, const DevMem2Df& src2, const DevMem2D_<short4>& dst, hipStream_t stream)
    {
        transform(static_cast< DevMem2D_<short4> >(src1), src2, static_cast< DevMem2D_<short4> >(dst), divide_16sc4_32f(), stream);
    }


    //////////////////////////////////////////////////////////////////////////
    // multiply

    template <> struct TransformFunctorTraits< plus<short> > : DefaultTransformFunctorTraits< plus<short> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T> void add_gpu(const DevMem2D src1, const DevMem2D src2, DevMem2D dst, hipStream_t stream)
    {
        transform((DevMem2D_<T>)src1, (DevMem2D_<T>)src2, (DevMem2D_<T>)dst, plus<T>(), stream);
    }

    template void add_gpu<short>(const DevMem2D src1, const DevMem2D src2, DevMem2D dst, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // multiply

    struct multiply_8uc4_32f : binary_function<uint, float, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, float b) const
        {
            uint res = 0;

            res |= (saturate_cast<uchar>((0xffu & (a      )) * b)      );
            res |= (saturate_cast<uchar>((0xffu & (a >>  8)) * b) <<  8);
            res |= (saturate_cast<uchar>((0xffu & (a >> 16)) * b) << 16);
            res |= (saturate_cast<uchar>((0xffu & (a >> 24)) * b) << 24);

            return res;
        }
    };

    template <> struct TransformFunctorTraits<multiply_8uc4_32f> : DefaultTransformFunctorTraits<multiply_8uc4_32f>
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void multiply_gpu(const DevMem2D_<uchar4>& src1, const DevMem2Df& src2, const DevMem2D_<uchar4>& dst, hipStream_t stream)
    {
        transform(static_cast< DevMem2D_<uint> >(src1), src2, static_cast< DevMem2D_<uint> >(dst), multiply_8uc4_32f(), stream);
    }

    struct multiply_16sc4_32f : binary_function<short4, float, short4>
    {
        __device__ __forceinline__ short4 operator ()(short4 a, float b) const
        {
            return make_short4(saturate_cast<short>(a.x * b), saturate_cast<short>(a.y * b),
                               saturate_cast<short>(a.z * b), saturate_cast<short>(a.w * b));
        }
    };

    template <> struct TransformFunctorTraits<multiply_16sc4_32f> : DefaultTransformFunctorTraits<multiply_16sc4_32f>
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    void multiply_gpu(const DevMem2D_<short4>& src1, const DevMem2Df& src2, const DevMem2D_<short4>& dst, hipStream_t stream)
    {
        transform(static_cast< DevMem2D_<short4> >(src1), src2, 
                  static_cast< DevMem2D_<short4> >(dst), multiply_16sc4_32f(), stream);
    }


    //////////////////////////////////////////////////////////////////////////
    // multiply (by scalar)

    template <typename T, typename D, typename S> struct MultiplyScalar : unary_function<T, D>
    {
        __host__ __device__ __forceinline__ MultiplyScalar(typename TypeTraits<S>::ParameterType scale_) : scale(scale_) {}

        __device__ __forceinline__ D operator ()(typename TypeTraits<T>::ParameterType a) const
        {
            return saturate_cast<D>(a * scale);
        }

        const S scale;
    };

    template <> struct TransformFunctorTraits< MultiplyScalar<uchar, uchar, float> > : DefaultTransformFunctorTraits< MultiplyScalar<uchar, uchar, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 8 };
    };

    template <typename T, typename D>
    void multiplyScalar_gpu(const DevMem2D& src, float scale, const DevMem2D& dst, hipStream_t stream)
    {
        transform(static_cast< DevMem2D_<T> >(src), static_cast< DevMem2D_<D> >(dst), MultiplyScalar<T, D, float>(scale), stream);
    }

    template void multiplyScalar_gpu<uchar, uchar>(const DevMem2D& src, float scale, const DevMem2D& dst, hipStream_t stream);


    //////////////////////////////////////////////////////////////////////////
    // addWeighted

    template <typename T1, typename T2, typename D> struct AddWeighted : binary_function<T1, T2, D>
    {
        __host__ __device__ __forceinline__ AddWeighted(double alpha_, double beta_, double gamma_) : alpha(alpha_), beta(beta_), gamma(gamma_) {}

        __device__ __forceinline__ D operator ()(typename TypeTraits<T1>::ParameterType a, typename TypeTraits<T2>::ParameterType b) const
        {
            return saturate_cast<D>(alpha * a + beta * b + gamma);
        }

        const double alpha;
        const double beta;
        const double gamma;
    };

    template <> struct TransformFunctorTraits< AddWeighted<ushort, ushort, ushort> > : DefaultTransformFunctorTraits< AddWeighted<ushort, ushort, ushort> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<ushort, ushort, short> > : DefaultTransformFunctorTraits< AddWeighted<ushort, ushort, short> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<ushort, short, ushort> > : DefaultTransformFunctorTraits< AddWeighted<ushort, short, ushort> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<ushort, short, short> > : DefaultTransformFunctorTraits< AddWeighted<ushort, short, short> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<short, short, ushort> > : DefaultTransformFunctorTraits< AddWeighted<short, short, ushort> >
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<short, short, short> > : DefaultTransformFunctorTraits< AddWeighted<short, short, short> >
    {
        enum { smart_shift = 4 };
    };

    template <> struct TransformFunctorTraits< AddWeighted<int, int, int> > : DefaultTransformFunctorTraits< AddWeighted<int, int, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<int, int, float> > : DefaultTransformFunctorTraits< AddWeighted<int, int, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<int, float, int> > : DefaultTransformFunctorTraits< AddWeighted<int, float, int> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<int, float, float> > : DefaultTransformFunctorTraits< AddWeighted<int, float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<float, float, int> > : DefaultTransformFunctorTraits< AddWeighted<float, float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits< AddWeighted<float, float, float> > : DefaultTransformFunctorTraits< AddWeighted<float, float, float> >
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <typename T1, typename T2, typename D>
    void addWeighted_gpu(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&alpha) );
        cudaSafeCall( cudaSetDoubleForDevice(&beta) );
        cudaSafeCall( cudaSetDoubleForDevice(&gamma) );

        AddWeighted<T1, T2, D> op(alpha, beta, gamma);

        transform(static_cast< DevMem2D_<T1> >(src1), static_cast< DevMem2D_<T2> >(src2), static_cast< DevMem2D_<D> >(dst), op, stream);
    }

    template void addWeighted_gpu<uchar, uchar, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, uchar, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, schar, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, schar, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, ushort, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, ushort, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, short, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, short, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, int, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, int, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, float, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, float, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<uchar, double, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<uchar, double, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);



    template void addWeighted_gpu<schar, schar, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, schar, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, ushort, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, ushort, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, short, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, short, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, int, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, int, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, float, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, float, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<schar, double, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<schar, double, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);



    template void addWeighted_gpu<ushort, ushort, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, ushort, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, short, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, short, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, int, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, int, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, float, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, float, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<ushort, double, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<ushort, double, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);



    template void addWeighted_gpu<short, short, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, short, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<short, int, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, int, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<short, float, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, float, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<short, double, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<short, double, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    

    template void addWeighted_gpu<int, int, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, int, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<int, float, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, float, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<int, double, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<int, double, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    

    template void addWeighted_gpu<float, float, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, float, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    template void addWeighted_gpu<float, double, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<float, double, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);

    

    template void addWeighted_gpu<double, double, uchar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, schar>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, ushort>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, short>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, int>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, float>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
    template void addWeighted_gpu<double, double, double>(const DevMem2D& src1, double alpha, const DevMem2D& src2, double beta, double gamma, const DevMem2D& dst, hipStream_t stream);
}}}
