#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <opencv2/gpu/device/lbp.hpp>

namespace cv { namespace gpu { namespace device
{
    namespace lbp
    {
        __global__ void lbp_classify(const DevMem2D_< ::cv::gpu::device::Stage> stages, const DevMem2Di trees, const DevMem2Db nodes, const DevMem2Df leaves, const DevMem2Di subsets,
            const DevMem2Db integral, float step, int subsetSize, DevMem2D_<int4> objects)
        {
            unsigned int x = threadIdx.x;
            unsigned int y = blockIdx.x;
            int nodeOfs = 0, leafOfs = 0;
            ::cv::gpu::device::Feature feature;

            for (int s = 0; s < stages.cols; s++ )
            {
                ::cv::gpu::device::Stage stage = stages(0, s);
                int sum = 0;
                for (int w = 0; w < stage.ntrees; w++)
                {
                    ::cv::gpu::device::ClNode node = nodes(0, nodeOfs);
                    char c = feature();// TODO: inmplement it
                    const int subsetIdx = (nodeOfs * subsetSize);
                    int idx = subsetIdx + ((c >> 5) & ( 1 << (c & 31)) ? leafOfs : leafOfs + 1);
                    sum += leaves(0, subsets(0, idx) );
                    nodeOfs++;
                    leafOfs += 2;
                }

                if (sum < stage.threshold)
                    return; // nothing matched
                return;//mathed
            }

        }

        void cascadeClassify(const DevMem2Db bstages, const DevMem2Di trees, const DevMem2Db nodes, const DevMem2Df leaves, const DevMem2Di subsets,
            const DevMem2Db integral, int workWidth, int workHeight, int step, int subsetSize, DevMem2D_<int4> objects, int minNeighbors, hipStream_t stream)
        {
            printf("CascadeClassify");
            int blocks = ceilf(workHeight / (float)step);
            int threads = ceilf(workWidth / (float)step);
            DevMem2D_< ::cv::gpu::device::Stage> stages = DevMem2D_< ::cv::gpu::device::Stage>(bstages);

            lbp_classify<<<blocks, threads>>>(stages, trees, nodes, leaves, subsets, integral, step, subsetSize, objects);
        }
    }
}}}