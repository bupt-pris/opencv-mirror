#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/vec_distance.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace cv { namespace gpu { namespace bf_radius_match
{
    template <typename T> struct SingleTrain
    {
        enum {USE_IMG_IDX = 0};

        explicit SingleTrain(const DevMem2D_<T>& train_) : train(train_)
        {
        }

        static __device__ __forceinline__ void store(const int* s_trainIdx, const int* s_imgIdx, const float* s_dist, unsigned int& s_count, int& s_globInd, 
            int* trainIdx, int* imgIdx, float* distance, int maxCount)
        {
            const int tid = threadIdx.y * blockDim.x + threadIdx.x;

            if (tid < s_count && s_globInd + tid < maxCount)
            {
                trainIdx[s_globInd + tid] = s_trainIdx[tid];
                distance[s_globInd + tid] = s_dist[tid];
            }

            if (tid == 0)
            {
                s_globInd += s_count;
                s_count = 0;
            }
        }

        template <int BLOCK_STACK, typename Dist, typename VecDiff, typename Mask>
        __device__ __forceinline__ void loop(float maxDistance, Mask& mask, const VecDiff& vecDiff, 
            int* s_trainIdx, int* s_imgIdx, float* s_dist, unsigned int& s_count, int& s_globInd, 
            int* trainIdxRow, int* imgIdxRow, float* distanceRow, int maxCount, 
            typename Dist::result_type* s_diffRow) const
        {
            #if __CUDA_ARCH__ >= 120

            for (int i = 0; i < train.rows; i += blockDim.y)
            {
                int trainIdx = i + threadIdx.y;

                if (trainIdx < train.rows && mask(blockIdx.x, trainIdx))
                {
                    Dist dist;
                    
                    vecDiff.calc(train.ptr(trainIdx), train.cols, dist, s_diffRow, threadIdx.x);

                    const typename Dist::result_type val = dist;

                    if (threadIdx.x == 0 && val < maxDistance)
                    {
                        unsigned int ind = atomicInc(&s_count, (unsigned int) -1);
                        s_trainIdx[ind] = trainIdx;
                        s_dist[ind] = val;
                    }
                }

                __syncthreads();

                if (s_count >= BLOCK_STACK - blockDim.y)
                    store(s_trainIdx, s_imgIdx, s_dist, s_count, s_globInd, trainIdxRow, imgIdxRow, distanceRow, maxCount);

                __syncthreads();
            }

            store(s_trainIdx, s_imgIdx, s_dist, s_count, s_globInd, trainIdxRow, imgIdxRow, distanceRow, maxCount);

            #endif
        }

        __device__ __forceinline__ int descLen() const
        {
            return train.cols;
        }

        const DevMem2D_<T> train;
    };

    template <typename T> struct TrainCollection
    {
        enum {USE_IMG_IDX = 1};

        TrainCollection(const DevMem2D_<T>* trainCollection_, int nImg_, int desclen_) : 
            trainCollection(trainCollection_), nImg(nImg_), desclen(desclen_)
        {
        }

        static __device__ __forceinline__ void store(const int* s_trainIdx, const int* s_imgIdx, const float* s_dist, unsigned int& s_count, int& s_globInd, 
            int* trainIdx, int* imgIdx, float* distance, int maxCount)
        {
            const int tid = threadIdx.y * blockDim.x + threadIdx.x;

            if (tid < s_count && s_globInd + tid < maxCount)
            {
                trainIdx[s_globInd + tid] = s_trainIdx[tid];
                imgIdx[s_globInd + tid] = s_imgIdx[tid];
                distance[s_globInd + tid] = s_dist[tid];
            }

            if (tid == 0)
            {
                s_globInd += s_count;
                s_count = 0;
            }
        }

        template <int BLOCK_STACK, typename Dist, typename VecDiff, typename Mask>
        __device__ void loop(float maxDistance, Mask& mask, const VecDiff& vecDiff, 
            int* s_trainIdx, int* s_imgIdx, float* s_dist, unsigned int& s_count, int& s_globInd, 
            int* trainIdxRow, int* imgIdxRow, float* distanceRow, int maxCount, 
            typename Dist::result_type* s_diffRow) const
        {
            #if __CUDA_ARCH__ >= 120

            for (int imgIdx = 0; imgIdx < nImg; ++imgIdx)
            {
                const DevMem2D_<T> train = trainCollection[imgIdx];

                mask.next();

                for (int i = 0; i < train.rows; i += blockDim.y)
                {
                    int trainIdx = i + threadIdx.y;

                    if (trainIdx < train.rows && mask(blockIdx.x, trainIdx))
                    {
                        Dist dist;
                        
                        vecDiff.calc(train.ptr(trainIdx), desclen, dist, s_diffRow, threadIdx.x);

                        const typename Dist::result_type val = dist;

                        if (threadIdx.x == 0 && val < maxDistance)
                        {
                            unsigned int ind = atomicInc(&s_count, (unsigned int) -1);
                            s_trainIdx[ind] = trainIdx;
                            s_imgIdx[ind] = imgIdx;
                            s_dist[ind] = val;
                        }
                    }

                    __syncthreads();

                    if (s_count >= BLOCK_STACK - blockDim.y)
                        store(s_trainIdx, s_imgIdx, s_dist, s_count, s_globInd, trainIdxRow, imgIdxRow, distanceRow, maxCount);

                    __syncthreads();
                }
            }

            store(s_trainIdx, s_imgIdx, s_dist, s_count, s_globInd, trainIdxRow, imgIdxRow, distanceRow, maxCount);

            #endif
        }

        __device__ __forceinline__ int descLen() const
        {
            return desclen;
        }

        const DevMem2D_<T>* trainCollection;
        const int nImg;
        const int desclen;
    };

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, int BLOCK_STACK, typename VecDiff, typename Dist, typename T, typename Train, typename Mask>
    __global__ void radiusMatch(const PtrStep_<T> query, const Train train, float maxDistance, const Mask mask, 
        PtrStepi trainIdx, PtrStepi imgIdx, PtrStepf distance, int* nMatches, int maxCount)
    {
        typedef typename Dist::result_type result_type;
        typedef typename Dist::value_type value_type;

        __shared__ result_type s_mem[BLOCK_DIM_X * BLOCK_DIM_Y];

        __shared__ int s_trainIdx[BLOCK_STACK];
        __shared__ int s_imgIdx[Train::USE_IMG_IDX ? BLOCK_STACK : 1];
        __shared__ float s_dist[BLOCK_STACK];
        __shared__ unsigned int s_count;

        __shared__ int s_globInd;

        if (threadIdx.x == 0 && threadIdx.y == 0)
        {
            s_count = 0;
            s_globInd = 0;
        }
        __syncthreads();

        const VecDiff vecDiff(query.ptr(blockIdx.x), train.descLen(), (typename Dist::value_type*)s_mem, threadIdx.y * BLOCK_DIM_X + threadIdx.x, threadIdx.x);

        Mask m = mask;

        train.template loop<BLOCK_STACK, Dist>(maxDistance, m, vecDiff, 
            s_trainIdx, s_imgIdx, s_dist, s_count, s_globInd, 
            trainIdx.ptr(blockIdx.x), imgIdx.ptr(blockIdx.x), distance.ptr(blockIdx.x), maxCount, 
            s_mem + BLOCK_DIM_X * threadIdx.y);

        if (threadIdx.x == 0 && threadIdx.y == 0)
            nMatches[blockIdx.x] = s_globInd;
    }

    ///////////////////////////////////////////////////////////////////////////////
    // Radius Match kernel caller

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, int BLOCK_STACK, typename Dist, typename T, typename Train, typename Mask>
    void radiusMatchSimple_caller(const DevMem2D_<T>& query, const Train& train, float maxDistance, const Mask& mask, 
        const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, int* nMatches,
        hipStream_t stream)
    {
        StaticAssert<BLOCK_STACK >= BLOCK_DIM_Y>::check();
        StaticAssert<BLOCK_STACK <= BLOCK_DIM_X * BLOCK_DIM_Y>::check();

        const dim3 grid(query.rows, 1, 1);
        const dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y, 1);

        radiusMatch<BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_STACK, VecDiffGlobal<BLOCK_DIM_X, T>, Dist, T>
            <<<grid, threads, 0, stream>>>(query, train, maxDistance, mask, trainIdx, imgIdx, distance, nMatches, trainIdx.cols);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, int BLOCK_STACK, int MAX_LEN, bool LEN_EQ_MAX_LEN, typename Dist, typename T, typename Train, typename Mask>
    void radiusMatchCached_caller(const DevMem2D_<T>& query, const Train& train, float maxDistance, const Mask& mask, 
        const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, int* nMatches, 
        hipStream_t stream)
    {
        StaticAssert<BLOCK_STACK >= BLOCK_DIM_Y>::check();
        StaticAssert<BLOCK_STACK <= BLOCK_DIM_X * BLOCK_DIM_Y>::check();
        StaticAssert<BLOCK_DIM_X * BLOCK_DIM_Y >= MAX_LEN>::check();
        StaticAssert<MAX_LEN % BLOCK_DIM_X == 0>::check();

        const dim3 grid(query.rows, 1, 1);
        const dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y, 1);

        radiusMatch<BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_STACK, VecDiffCachedRegister<BLOCK_DIM_X, MAX_LEN, LEN_EQ_MAX_LEN, typename Dist::value_type>, Dist, T>
              <<<grid, threads, 0, stream>>>(query, train, maxDistance, mask, trainIdx, imgIdx, distance, nMatches, trainIdx.cols);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    ///////////////////////////////////////////////////////////////////////////////
    // Radius Match Dispatcher
    
    template <typename Dist, typename T, typename Train, typename Mask>
    void radiusMatchDispatcher(const DevMem2D_<T>& query, const Train& train, float maxDistance, const Mask& mask, 
        const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, 
        hipStream_t stream)
    {
        if (query.cols < 64)
        {
            radiusMatchCached_caller<16, 16, 64, 64, false, Dist>(
                query, train, maxDistance, mask, 
                static_cast<DevMem2Di>(trainIdx), static_cast<DevMem2Di>(imgIdx), static_cast<DevMem2Df>(distance), (int*)nMatches.data,
                stream);
        }
        else if (query.cols == 64)
        {
            radiusMatchCached_caller<16, 16, 64, 64, true, Dist>(
                query, train, maxDistance, mask, 
                static_cast<DevMem2Di>(trainIdx), static_cast<DevMem2Di>(imgIdx), static_cast<DevMem2Df>(distance), (int*)nMatches.data,
                stream);
        }
        else if (query.cols < 128)
        {
            radiusMatchCached_caller<16, 16, 64, 128, false, Dist>(
                query, train, maxDistance, mask, 
                static_cast<DevMem2Di>(trainIdx), static_cast<DevMem2Di>(imgIdx), static_cast<DevMem2Df>(distance), (int*)nMatches.data,
                stream);
        }
        else if (query.cols == 128)
        {
            radiusMatchCached_caller<16, 16, 64, 128, true, Dist>(
                query, train, maxDistance, mask, 
                static_cast<DevMem2Di>(trainIdx), static_cast<DevMem2Di>(imgIdx), static_cast<DevMem2Df>(distance), (int*)nMatches.data,
                stream);
        }
        else if (query.cols < 256)
        {
            radiusMatchCached_caller<16, 16, 64, 256, false, Dist>(
                query, train, maxDistance, mask, 
                static_cast<DevMem2Di>(trainIdx), static_cast<DevMem2Di>(imgIdx), static_cast<DevMem2Df>(distance), (int*)nMatches.data,
                stream);
        }
        else if (query.cols == 256)
        {
            radiusMatchCached_caller<16, 16, 64, 256, true, Dist>(
                query, train, maxDistance, mask, 
                static_cast<DevMem2Di>(trainIdx), static_cast<DevMem2Di>(imgIdx), static_cast<DevMem2Df>(distance), (int*)nMatches.data, 
                stream);
        }
        else
        {
            radiusMatchSimple_caller<16, 16, 64, Dist>(
                query, train, maxDistance, mask, 
                static_cast<DevMem2Di>(trainIdx), static_cast<DevMem2Di>(imgIdx), static_cast<DevMem2Df>(distance), (int*)nMatches.data,
                stream);
        }
    }    
    
    ///////////////////////////////////////////////////////////////////////////////
    // Radius Match caller

    template <typename T> void radiusMatchSingleL1_gpu(const DevMem2D& query, const DevMem2D& train_, float maxDistance, const DevMem2D& mask, 
        const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, 
        hipStream_t stream)
    {
        SingleTrain<T> train(static_cast< DevMem2D_<T> >(train_));

        if (mask.data)
        {
            radiusMatchDispatcher< L1Dist<T> >(static_cast< DevMem2D_<T> >(query), train, maxDistance, SingleMask(mask), 
                trainIdx, DevMem2D(), distance, nMatches, 
                stream);
        }
        else
        {
            radiusMatchDispatcher< L1Dist<T> >(static_cast< DevMem2D_<T> >(query), train, maxDistance, WithOutMask(), 
                trainIdx, DevMem2D(), distance, nMatches, 
                stream);
        }
    }

    template void radiusMatchSingleL1_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchSingleL1_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchSingleL1_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchSingleL1_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchSingleL1_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchSingleL1_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);

    template <typename T> void radiusMatchSingleL2_gpu(const DevMem2D& query, const DevMem2D& train_, float maxDistance, const DevMem2D& mask, 
        const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, 
        hipStream_t stream)
    {
        SingleTrain<T> train(static_cast< DevMem2D_<T> >(train_));

        if (mask.data)
        {
            radiusMatchDispatcher<L2Dist>(static_cast< DevMem2D_<T> >(query), train, maxDistance, SingleMask(mask), 
                trainIdx, DevMem2D(), distance, nMatches, 
                stream);
        }
        else
        {
            radiusMatchDispatcher<L2Dist>(static_cast< DevMem2D_<T> >(query), train, maxDistance, WithOutMask(), 
                trainIdx, DevMem2D(), distance, nMatches, 
                stream);
        }
    }

    //template void radiusMatchSingleL2_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchSingleL2_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchSingleL2_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchSingleL2_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchSingleL2_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchSingleL2_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);

    template <typename T> void radiusMatchSingleHamming_gpu(const DevMem2D& query, const DevMem2D& train_, float maxDistance, const DevMem2D& mask, 
        const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, 
        hipStream_t stream)
    {
        SingleTrain<T> train(static_cast< DevMem2D_<T> >(train_));

        if (mask.data)
        {
            radiusMatchDispatcher<HammingDist>(static_cast< DevMem2D_<T> >(query), train, maxDistance, SingleMask(mask), 
                trainIdx, DevMem2D(), distance, nMatches, 
                stream);
        }
        else
        {
            radiusMatchDispatcher<HammingDist>(static_cast< DevMem2D_<T> >(query), train, maxDistance, WithOutMask(), 
                trainIdx, DevMem2D(), distance, nMatches, 
                stream);
        }
    }

    template void radiusMatchSingleHamming_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchSingleHamming_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchSingleHamming_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchSingleHamming_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchSingleHamming_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);

    template <typename T> void radiusMatchCollectionL1_gpu(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, 
        const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, 
        hipStream_t stream)
    {
        TrainCollection<T> train((DevMem2D_<T>*)trainCollection.ptr(), trainCollection.cols, query.cols);

        if (maskCollection.data)
        {
            radiusMatchDispatcher< L1Dist<T> >(static_cast< DevMem2D_<T> >(query), train, maxDistance, MaskCollection(maskCollection.data), 
                trainIdx, imgIdx, distance, nMatches, 
                stream);
        }
        else
        {
            radiusMatchDispatcher< L1Dist<T> >(static_cast< DevMem2D_<T> >(query), train, maxDistance, WithOutMask(), 
                trainIdx, imgIdx, distance, nMatches, 
                stream);
        }
    }

    template void radiusMatchCollectionL1_gpu<uchar >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchCollectionL1_gpu<schar >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchCollectionL1_gpu<ushort>(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchCollectionL1_gpu<short >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchCollectionL1_gpu<int   >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchCollectionL1_gpu<float >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);

    template <typename T> void radiusMatchCollectionL2_gpu(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, 
        const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, 
        hipStream_t stream)
    {
        TrainCollection<T> train((DevMem2D_<T>*)trainCollection.ptr(), trainCollection.cols, query.cols);

        if (maskCollection.data)
        {
            radiusMatchDispatcher<L2Dist>(static_cast< DevMem2D_<T> >(query), train, maxDistance, MaskCollection(maskCollection.data), 
                trainIdx, imgIdx, distance, nMatches, 
                stream);
        }
        else
        {
            radiusMatchDispatcher<L2Dist>(static_cast< DevMem2D_<T> >(query), train, maxDistance, WithOutMask(), 
                trainIdx, imgIdx, distance, nMatches, 
                stream);
        }
    }

    //template void radiusMatchCollectionL2_gpu<uchar >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchCollectionL2_gpu<schar >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchCollectionL2_gpu<ushort>(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchCollectionL2_gpu<short >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchCollectionL2_gpu<int   >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchCollectionL2_gpu<float >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);

    template <typename T> void radiusMatchCollectionHamming_gpu(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, 
        const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, 
        hipStream_t stream)
    {
        TrainCollection<T> train((DevMem2D_<T>*)trainCollection.ptr(), trainCollection.cols, query.cols);

        if (maskCollection.data)
        {
            radiusMatchDispatcher<HammingDist>(static_cast< DevMem2D_<T> >(query), train, maxDistance, MaskCollection(maskCollection.data), 
                trainIdx, imgIdx, distance, nMatches, 
                stream);
        }
        else
        {
            radiusMatchDispatcher<HammingDist>(static_cast< DevMem2D_<T> >(query), train, maxDistance, WithOutMask(), 
                trainIdx, imgIdx, distance, nMatches, 
                stream);
        }
    }

    template void radiusMatchCollectionHamming_gpu<uchar >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchCollectionHamming_gpu<schar >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchCollectionHamming_gpu<ushort>(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    //template void radiusMatchCollectionHamming_gpu<short >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
    template void radiusMatchCollectionHamming_gpu<int   >(const DevMem2D& query, const DevMem2D& trainCollection, float maxDistance, const DevMem2D_<PtrStep>& maskCollection, const DevMem2D& trainIdx, const DevMem2D& imgIdx, const DevMem2D& distance, const DevMem2D& nMatches, hipStream_t stream);
}}}
