#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"

namespace cv { namespace gpu { namespace device 
{
    namespace imgproc 
    {
        template <typename T, typename B> __global__ void pyrDown(const PtrStep<T> src, PtrStep<T> dst, const B b, int dst_cols)
        {
            typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type value_type;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y;

            __shared__ value_type smem[256 + 4];

            value_type sum;
            
            const int src_y = 2*y;

            sum = VecTraits<value_type>::all(0);
            
            sum = sum + 0.0625f * b.at(src_y - 2, x, src.data, src.step);
            sum = sum + 0.25f   * b.at(src_y - 1, x, src.data, src.step);
            sum = sum + 0.375f  * b.at(src_y    , x, src.data, src.step);
            sum = sum + 0.25f   * b.at(src_y + 1, x, src.data, src.step);
            sum = sum + 0.0625f * b.at(src_y + 2, x, src.data, src.step);

            smem[2 + threadIdx.x] = sum;

            if (threadIdx.x < 2)
            {
                const int left_x = x - 2 + threadIdx.x;

                sum = VecTraits<value_type>::all(0);
            
                sum = sum + 0.0625f * b.at(src_y - 2, left_x, src.data, src.step);
                sum = sum + 0.25f   * b.at(src_y - 1, left_x, src.data, src.step);
                sum = sum + 0.375f  * b.at(src_y    , left_x, src.data, src.step);
                sum = sum + 0.25f   * b.at(src_y + 1, left_x, src.data, src.step);
                sum = sum + 0.0625f * b.at(src_y + 2, left_x, src.data, src.step);

                smem[threadIdx.x] = sum;
            }

            if (threadIdx.x > 253)
            {
                const int right_x = x + threadIdx.x + 2;

                sum = VecTraits<value_type>::all(0);
            
                sum = sum + 0.0625f * b.at(src_y - 2, right_x, src.data, src.step);
                sum = sum + 0.25f   * b.at(src_y - 1, right_x, src.data, src.step);
                sum = sum + 0.375f  * b.at(src_y    , right_x, src.data, src.step);
                sum = sum + 0.25f   * b.at(src_y + 1, right_x, src.data, src.step);
                sum = sum + 0.0625f * b.at(src_y + 2, right_x, src.data, src.step);

                smem[4 + threadIdx.x] = sum;
            }

            __syncthreads();

            if (threadIdx.x < 128)
            {
                const int tid2 = threadIdx.x * 2;

                sum = VecTraits<value_type>::all(0);

                sum = sum + 0.0625f * smem[2 + tid2 - 2];
                sum = sum + 0.25f   * smem[2 + tid2 - 1];
                sum = sum + 0.375f  * smem[2 + tid2    ];
                sum = sum + 0.25f   * smem[2 + tid2 + 1];
                sum = sum + 0.0625f * smem[2 + tid2 + 2];

                const int dst_x = (blockIdx.x * blockDim.x + tid2) / 2;

                if (dst_x < dst_cols)
                    dst.ptr(y)[dst_x] = saturate_cast<T>(sum);
            }
        }

        template <typename T, template <typename> class B> void pyrDown_caller(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, hipStream_t stream)
        {
            const dim3 block(256);
            const dim3 grid(divUp(src.cols, block.x), dst.rows);

            B<T> b(src.rows, src.cols);

            pyrDown<T><<<grid, block, 0, stream>>>(src, dst, b, dst.cols);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <typename T, int cn> void pyrDown_gpu(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream)
        {
            typedef typename TypeVec<T, cn>::vec_type type;

            typedef void (*caller_t)(const DevMem2D_<type>& src, const DevMem2D_<type>& dst, hipStream_t stream);

            static const caller_t callers[] = 
            {
                pyrDown_caller<type, BrdReflect101>, pyrDown_caller<type, BrdReplicate>, pyrDown_caller<type, BrdConstant>, pyrDown_caller<type, BrdReflect>, pyrDown_caller<type, BrdWrap>
            };

            callers[borderType](static_cast< DevMem2D_<type> >(src), static_cast< DevMem2D_<type> >(dst), stream);
        }

        template void pyrDown_gpu<uchar, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<uchar, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<uchar, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<uchar, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

        template void pyrDown_gpu<schar, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<schar, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<schar, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<schar, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

        template void pyrDown_gpu<ushort, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<ushort, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<ushort, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<ushort, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

        template void pyrDown_gpu<short, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<short, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<short, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<short, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

        template void pyrDown_gpu<int, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<int, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<int, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<int, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

        template void pyrDown_gpu<float, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<float, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<float, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrDown_gpu<float, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
    } // namespace imgproc
}}} // namespace cv { namespace gpu { namespace device
