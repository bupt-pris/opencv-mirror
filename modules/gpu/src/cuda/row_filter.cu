#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Copyright (C) 1993-2011, NVIDIA Corporation, all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

#define MAX_KERNEL_SIZE 16
#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 4
#define RESULT_STEPS 8
#define HALO_STEPS 1

namespace filter_row
{
    __constant__ float c_kernel[MAX_KERNEL_SIZE];

    void loadKernel(const float kernel[], int ksize)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_kernel), kernel, ksize * sizeof(float)) );
    }

    namespace detail
    {
        template <typename T, size_t size> struct SmemType
        {
            typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type smem_t;
        };

        template <typename T> struct SmemType<T, 4>
        {
            typedef T smem_t;
        };
    }

    template <typename T> struct SmemType
    {
        typedef typename detail::SmemType<T, sizeof(T)>::smem_t smem_t;
    };

    template <int KERNEL_SIZE, typename T, typename D, typename B>
    __global__ void linearRowFilter(const DevMem2D_<T> src, PtrStep<D> dst, int anchor, const B b)
    {
        typedef typename SmemType<T>::smem_t smem_t;
        typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type sum_t;

        __shared__ smem_t smem[BLOCK_DIM_Y][(RESULT_STEPS + 2 * HALO_STEPS) * BLOCK_DIM_X];

        //Offset to the left halo edge
        const int x = (blockIdx.x * RESULT_STEPS - HALO_STEPS) * BLOCK_DIM_X + threadIdx.x;
        const int y = blockIdx.y * BLOCK_DIM_Y + threadIdx.y;

        if (y < src.rows)
        {
            const T* src_row = src.ptr(y);

            //Load main data
            #pragma unroll
            for(int i = HALO_STEPS; i < HALO_STEPS + RESULT_STEPS; ++i)
                smem[threadIdx.y][threadIdx.x + i * BLOCK_DIM_X] = b.at_high(i * BLOCK_DIM_X + x, src_row);

            //Load left halo
            #pragma unroll
            for(int i = 0; i < HALO_STEPS; ++i)
                smem[threadIdx.y][threadIdx.x + i * BLOCK_DIM_X] = b.at_low(i * BLOCK_DIM_X + x, src_row);

            //Load right halo
            #pragma unroll
            for(int i = HALO_STEPS + RESULT_STEPS; i < HALO_STEPS + RESULT_STEPS + HALO_STEPS; ++i)
                smem[threadIdx.y][threadIdx.x + i * BLOCK_DIM_X] = b.at_high(i * BLOCK_DIM_X + x, src_row);

            __syncthreads();

            D* dst_row = dst.ptr(y);

            #pragma unroll
            for(int i = HALO_STEPS; i < HALO_STEPS + RESULT_STEPS; ++i)
            {
                sum_t sum = VecTraits<sum_t>::all(0);

                #pragma unroll
                for (int j = 0; j < KERNEL_SIZE; ++j)
                    sum = sum + smem[threadIdx.y][threadIdx.x + i * BLOCK_DIM_X + j - anchor] * c_kernel[j];

                int dstX = x + i * BLOCK_DIM_X;

                if (dstX < src.cols)
                    dst_row[dstX] = saturate_cast<D>(sum);
            }
        }
    }
}

namespace cv { namespace gpu { namespace filters
{
    template <int ksize, typename T, typename D, template<typename> class B>
    void linearRowFilter_caller(const DevMem2D_<T>& src, const DevMem2D_<D>& dst, int anchor, hipStream_t stream)
    {
        typedef typename filter_row::SmemType<T>::smem_t smem_t;

        const dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
        const dim3 grid(divUp(src.cols, RESULT_STEPS * BLOCK_DIM_X), divUp(src.rows, BLOCK_DIM_Y));

        B<smem_t> b(src.cols);

        filter_row::linearRowFilter<ksize, T, D><<<grid, block, 0, stream>>>(src, dst, anchor, b);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <typename T, typename D>
    void linearRowFilter_gpu(const DevMem2Db& src, const DevMem2Db& dst, const float kernel[], int ksize, int anchor, int brd_type, hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2D_<T>& src, const DevMem2D_<D>& dst, int anchor, hipStream_t stream);
        static const caller_t callers[5][17] = 
        {
            {
                0, 
                linearRowFilter_caller<1 , T, D, BrdRowReflect101>, 
                linearRowFilter_caller<2 , T, D, BrdRowReflect101>,
                linearRowFilter_caller<3 , T, D, BrdRowReflect101>, 
                linearRowFilter_caller<4 , T, D, BrdRowReflect101>, 
                linearRowFilter_caller<5 , T, D, BrdRowReflect101>, 
                linearRowFilter_caller<6 , T, D, BrdRowReflect101>, 
                linearRowFilter_caller<7 , T, D, BrdRowReflect101>,
                linearRowFilter_caller<8 , T, D, BrdRowReflect101>,
                linearRowFilter_caller<9 , T, D, BrdRowReflect101>, 
                linearRowFilter_caller<10, T, D, BrdRowReflect101>, 
                linearRowFilter_caller<11, T, D, BrdRowReflect101>, 
                linearRowFilter_caller<12, T, D, BrdRowReflect101>, 
                linearRowFilter_caller<13, T, D, BrdRowReflect101>, 
                linearRowFilter_caller<14, T, D, BrdRowReflect101>,
                linearRowFilter_caller<15, T, D, BrdRowReflect101>, 
                linearRowFilter_caller<16, T, D, BrdRowReflect101>
            },
            {
                0, 
                linearRowFilter_caller<1 , T, D, BrdRowReplicate>, 
                linearRowFilter_caller<2 , T, D, BrdRowReplicate>,
                linearRowFilter_caller<3 , T, D, BrdRowReplicate>, 
                linearRowFilter_caller<4 , T, D, BrdRowReplicate>, 
                linearRowFilter_caller<5 , T, D, BrdRowReplicate>, 
                linearRowFilter_caller<6 , T, D, BrdRowReplicate>, 
                linearRowFilter_caller<7 , T, D, BrdRowReplicate>, 
                linearRowFilter_caller<8 , T, D, BrdRowReplicate>,
                linearRowFilter_caller<9 , T, D, BrdRowReplicate>, 
                linearRowFilter_caller<10, T, D, BrdRowReplicate>, 
                linearRowFilter_caller<11, T, D, BrdRowReplicate>, 
                linearRowFilter_caller<12, T, D, BrdRowReplicate>, 
                linearRowFilter_caller<13, T, D, BrdRowReplicate>, 
                linearRowFilter_caller<14, T, D, BrdRowReplicate>,
                linearRowFilter_caller<15, T, D, BrdRowReplicate>, 
                linearRowFilter_caller<16, T, D, BrdRowReplicate>
            },
            {
                0, 
                linearRowFilter_caller<1 , T, D, BrdRowConstant>, 
                linearRowFilter_caller<2 , T, D, BrdRowConstant>,
                linearRowFilter_caller<3 , T, D, BrdRowConstant>, 
                linearRowFilter_caller<4 , T, D, BrdRowConstant>, 
                linearRowFilter_caller<5 , T, D, BrdRowConstant>, 
                linearRowFilter_caller<6 , T, D, BrdRowConstant>, 
                linearRowFilter_caller<7 , T, D, BrdRowConstant>, 
                linearRowFilter_caller<8 , T, D, BrdRowConstant>,
                linearRowFilter_caller<9 , T, D, BrdRowConstant>,
                linearRowFilter_caller<10, T, D, BrdRowConstant>, 
                linearRowFilter_caller<11, T, D, BrdRowConstant>, 
                linearRowFilter_caller<12, T, D, BrdRowConstant>, 
                linearRowFilter_caller<13, T, D, BrdRowConstant>,
                linearRowFilter_caller<14, T, D, BrdRowConstant>,
                linearRowFilter_caller<15, T, D, BrdRowConstant>, 
                linearRowFilter_caller<16, T, D, BrdRowConstant>
            },
            {
                0, 
                linearRowFilter_caller<1 , T, D, BrdRowReflect>, 
                linearRowFilter_caller<2 , T, D, BrdRowReflect>,
                linearRowFilter_caller<3 , T, D, BrdRowReflect>, 
                linearRowFilter_caller<4 , T, D, BrdRowReflect>, 
                linearRowFilter_caller<5 , T, D, BrdRowReflect>, 
                linearRowFilter_caller<6 , T, D, BrdRowReflect>, 
                linearRowFilter_caller<7 , T, D, BrdRowReflect>, 
                linearRowFilter_caller<8 , T, D, BrdRowReflect>,
                linearRowFilter_caller<9 , T, D, BrdRowReflect>,
                linearRowFilter_caller<10, T, D, BrdRowReflect>, 
                linearRowFilter_caller<11, T, D, BrdRowReflect>, 
                linearRowFilter_caller<12, T, D, BrdRowReflect>, 
                linearRowFilter_caller<13, T, D, BrdRowReflect>,
                linearRowFilter_caller<14, T, D, BrdRowReflect>,
                linearRowFilter_caller<15, T, D, BrdRowReflect>, 
                linearRowFilter_caller<16, T, D, BrdRowReflect>
            },
            {
                0, 
                linearRowFilter_caller<1 , T, D, BrdRowWrap>, 
                linearRowFilter_caller<2 , T, D, BrdRowWrap>,
                linearRowFilter_caller<3 , T, D, BrdRowWrap>, 
                linearRowFilter_caller<4 , T, D, BrdRowWrap>, 
                linearRowFilter_caller<5 , T, D, BrdRowWrap>, 
                linearRowFilter_caller<6 , T, D, BrdRowWrap>, 
                linearRowFilter_caller<7 , T, D, BrdRowWrap>, 
                linearRowFilter_caller<8 , T, D, BrdRowWrap>,
                linearRowFilter_caller<9 , T, D, BrdRowWrap>,
                linearRowFilter_caller<10, T, D, BrdRowWrap>, 
                linearRowFilter_caller<11, T, D, BrdRowWrap>, 
                linearRowFilter_caller<12, T, D, BrdRowWrap>, 
                linearRowFilter_caller<13, T, D, BrdRowWrap>,
                linearRowFilter_caller<14, T, D, BrdRowWrap>,
                linearRowFilter_caller<15, T, D, BrdRowWrap>, 
                linearRowFilter_caller<16, T, D, BrdRowWrap>
            }
        };
        
        filter_row::loadKernel(kernel, ksize);

        callers[brd_type][ksize]((DevMem2D_<T>)src, (DevMem2D_<D>)dst, anchor, stream);
    }

    template void linearRowFilter_gpu<uchar , float >(const DevMem2Db& src, const DevMem2Db& dst, const float kernel[], int ksize, int anchor, int brd_type, hipStream_t stream);
    template void linearRowFilter_gpu<uchar4, float4>(const DevMem2Db& src, const DevMem2Db& dst, const float kernel[], int ksize, int anchor, int brd_type, hipStream_t stream);
    //template void linearRowFilter_gpu<short , float >(const DevMem2Db& src, const DevMem2Db& dst, const float kernel[], int ksize, int anchor, int brd_type, hipStream_t stream);
    //template void linearRowFilter_gpu<short2, float2>(const DevMem2Db& src, const DevMem2Db& dst, const float kernel[], int ksize, int anchor, int brd_type, hipStream_t stream);
    template void linearRowFilter_gpu<short3, float3>(const DevMem2Db& src, const DevMem2Db& dst, const float kernel[], int ksize, int anchor, int brd_type, hipStream_t stream);
    template void linearRowFilter_gpu<int   , float >(const DevMem2Db& src, const DevMem2Db& dst, const float kernel[], int ksize, int anchor, int brd_type, hipStream_t stream);
    template void linearRowFilter_gpu<float , float >(const DevMem2Db& src, const DevMem2Db& dst, const float kernel[], int ksize, int anchor, int brd_type, hipStream_t stream);
}}}
