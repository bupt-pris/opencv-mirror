#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"

BEGIN_OPENCV_DEVICE_NAMESPACE

namespace pyr_up {

template <typename T, typename B> __global__ void pyrUp(const PtrStep<T> src, DevMem2D_<T> dst, const B b)
{
    typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type value_type;

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ T smem1[10][10];
    __shared__ value_type smem2[20][16];

    value_type sum;

    if (threadIdx.x < 10 && threadIdx.y < 10)
        smem1[threadIdx.y][threadIdx.x] = b.at(blockIdx.y * blockDim.y / 2 + threadIdx.y - 1, blockIdx.x * blockDim.x / 2 + threadIdx.x - 1, src.data, src.step);

    __syncthreads();

    const int tidx = threadIdx.x;

    sum = VecTraits<value_type>::all(0);

    sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[1 + threadIdx.y / 2][1 + ((tidx - 2) >> 1)];
    sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[1 + threadIdx.y / 2][1 + ((tidx - 1) >> 1)];
    sum = sum + (tidx % 2 == 0) * 0.375f  * smem1[1 + threadIdx.y / 2][1 + ((tidx    ) >> 1)];
    sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[1 + threadIdx.y / 2][1 + ((tidx + 1) >> 1)];
    sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[1 + threadIdx.y / 2][1 + ((tidx + 2) >> 1)];

    smem2[2 + threadIdx.y][tidx] = sum;

    if (threadIdx.y < 2)
    {
        sum = VecTraits<value_type>::all(0);

        sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[0][1 + ((tidx - 2) >> 1)];
        sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[0][1 + ((tidx - 1) >> 1)];
        sum = sum + (tidx % 2 == 0) * 0.375f  * smem1[0][1 + ((tidx    ) >> 1)];
        sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[0][1 + ((tidx + 1) >> 1)];
        sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[0][1 + ((tidx + 2) >> 1)];

        smem2[threadIdx.y][tidx] = sum;
    }

    if (threadIdx.y > 13)
    {
        sum = VecTraits<value_type>::all(0);

        sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[9][1 + ((tidx - 2) >> 1)];
        sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[9][1 + ((tidx - 1) >> 1)];
        sum = sum + (tidx % 2 == 0) * 0.375f  * smem1[9][1 + ((tidx    ) >> 1)];
        sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[9][1 + ((tidx + 1) >> 1)];
        sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[9][1 + ((tidx + 2) >> 1)];

        smem2[4 + threadIdx.y][tidx] = sum;
    }

    __syncthreads();

    sum = VecTraits<value_type>::all(0);

    sum = sum + (tidx % 2 == 0) * 0.0625f * smem2[2 + threadIdx.y - 2][tidx];
    sum = sum + (tidx % 2 != 0) * 0.25f   * smem2[2 + threadIdx.y - 1][tidx];
    sum = sum + (tidx % 2 == 0) * 0.375f  * smem2[2 + threadIdx.y    ][tidx];
    sum = sum + (tidx % 2 != 0) * 0.25f   * smem2[2 + threadIdx.y + 1][tidx];
    sum = sum + (tidx % 2 == 0) * 0.0625f * smem2[2 + threadIdx.y + 2][tidx];

    if (x < dst.cols && y < dst.rows)
        dst.ptr(y)[x] = saturate_cast<T>(4.0f * sum);
}

template <typename T, template <typename> class B> void pyrUp_caller(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, hipStream_t stream)
{
    const dim3 block(16, 16);
    const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    B<T> b(src.rows, src.cols);

    pyrUp<T><<<grid, block, 0, stream>>>(src, dst, b);
    cudaSafeCall( hipGetLastError() );

    if (stream == 0)
        cudaSafeCall( hipDeviceSynchronize() );
}

template <typename T, int cn> void pyrUp_gpu(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream)
{
    typedef typename TypeVec<T, cn>::vec_type type;

    typedef void (*caller_t)(const DevMem2D_<type>& src, const DevMem2D_<type>& dst, hipStream_t stream);

    static const caller_t callers[] = 
    {
        pyrUp_caller<type, BrdReflect101>, pyrUp_caller<type, BrdReplicate>, pyrUp_caller<type, BrdConstant>, pyrUp_caller<type, BrdReflect>, pyrUp_caller<type, BrdWrap>
    };

    callers[borderType](static_cast< DevMem2D_<type> >(src), static_cast< DevMem2D_<type> >(dst), stream);
}

template void pyrUp_gpu<uchar, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<uchar, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<uchar, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<uchar, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

template void pyrUp_gpu<schar, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<schar, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<schar, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<schar, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

template void pyrUp_gpu<ushort, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<ushort, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<ushort, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<ushort, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

template void pyrUp_gpu<short, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<short, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<short, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<short, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

template void pyrUp_gpu<int, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<int, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<int, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<int, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

template void pyrUp_gpu<float, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<float, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<float, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
template void pyrUp_gpu<float, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

} // namespace pyr_up

END_OPENCV_DEVICE_NAMESPACE
