#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/vec_distance.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace cv { namespace gpu { namespace bfmatcher
{
    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename Dist, typename T, typename Mask>
    __global__ void radiusMatch(const PtrStep_<T> query, const DevMem2D_<T> train, float maxDistance, const Mask mask, 
        DevMem2Di trainIdx_, unsigned int* nMatches, PtrStepf distance)
    {
        #if __CUDA_ARCH__ >= 110

        __shared__ typename Dist::result_type smem[BLOCK_DIM_X * BLOCK_DIM_Y];

        typename Dist::result_type* sdiff_row = smem + BLOCK_DIM_X * threadIdx.y;
        
        const int queryIdx = blockIdx.x;
        const T* queryDescs = query.ptr(queryIdx);

        const int trainIdx = blockIdx.y * BLOCK_DIM_Y + threadIdx.y;

        if (trainIdx < train.rows)
        {
            const T* trainDescs = train.ptr(trainIdx);

            if (mask(queryIdx, trainIdx))
            {
                Dist dist;

                calcVecDiffGlobal<BLOCK_DIM_X>(queryDescs, trainDescs, train.cols, dist, sdiff_row, threadIdx.x);

                if (threadIdx.x == 0)
                {
                    if (dist < maxDistance)
                    {
                        unsigned int i = atomicInc(nMatches + queryIdx, (unsigned int) -1);
                        if (i < trainIdx_.cols)
                        {
                            distance.ptr(queryIdx)[i] = dist;
                            trainIdx_.ptr(queryIdx)[i] = trainIdx;
                        }
                    }
                }
            }
        }

        #endif
    }
        
    ///////////////////////////////////////////////////////////////////////////////
    // Radius Match kernel caller

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename Dist, typename T, typename Mask>
    void radiusMatch_caller(const DevMem2D_<T>& query, const DevMem2D_<T>& train, float maxDistance, const Mask& mask, 
        const DevMem2Di& trainIdx, const DevMem2D_<unsigned int>& nMatches, const DevMem2Df& distance, 
        hipStream_t stream)
    {
        const dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
        const dim3 grid(query.rows, divUp(train.rows, BLOCK_DIM_Y), 1);

        radiusMatch<BLOCK_DIM_X, BLOCK_DIM_Y, Dist, T><<<grid, threads, 0, stream>>>(query, train, maxDistance, mask, trainIdx, nMatches.data, distance);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // Radius Match Dispatcher

    template <typename Dist, typename T, typename Mask>
    void radiusMatchDispatcher(const DevMem2D_<T>& query, const DevMem2D_<T>& train, float maxDistance, const Mask& mask, 
        const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, 
        hipStream_t stream)
    {
        radiusMatch_caller<16, 16, Dist>(query, train, maxDistance, mask, 
            static_cast<DevMem2Di>(trainIdx), static_cast< const DevMem2D_<unsigned int> >(nMatches), static_cast<DevMem2Df>(distance), 
            stream);
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // Radius Match caller

    template <typename T> void radiusMatchL1_gpu(const DevMem2D& query, const DevMem2D& train, float maxDistance, const DevMem2D& mask, 
        const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, 
        hipStream_t stream)
    {
        if (mask.data)
        {
            radiusMatchDispatcher< L1Dist<T> >(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, SingleMask(mask), 
                trainIdx, nMatches, distance, 
                stream);
        }
        else
        {
            radiusMatchDispatcher< L1Dist<T> >(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, WithOutMask(), 
                trainIdx, nMatches, distance, 
                stream);
        }
    }

    template void radiusMatchL1_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    //template void radiusMatchL1_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    template void radiusMatchL1_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    template void radiusMatchL1_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    template void radiusMatchL1_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    template void radiusMatchL1_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);

    template <typename T> void radiusMatchL2_gpu(const DevMem2D& query, const DevMem2D& train, float maxDistance, const DevMem2D& mask, 
        const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, 
        hipStream_t stream)
    {
        if (mask.data)
        {
            radiusMatchDispatcher<L2Dist>(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, SingleMask(mask), 
                trainIdx, nMatches, distance, 
                stream);
        }
        else
        {
            radiusMatchDispatcher<L2Dist>(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, WithOutMask(), 
                trainIdx, nMatches, distance, 
                stream);
        }
    }

    //template void radiusMatchL2_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    //template void radiusMatchL2_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    //template void radiusMatchL2_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    //template void radiusMatchL2_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    //template void radiusMatchL2_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    template void radiusMatchL2_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);

    template <typename T> void radiusMatchHamming_gpu(const DevMem2D& query, const DevMem2D& train, float maxDistance, const DevMem2D& mask, 
        const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, 
        hipStream_t stream)
    {
        if (mask.data)
        {
            radiusMatchDispatcher<HammingDist>(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, SingleMask(mask), 
                trainIdx, nMatches, distance, 
                stream);
        }
        else
        {
            radiusMatchDispatcher<HammingDist>(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), maxDistance, WithOutMask(), 
                trainIdx, nMatches, distance, 
                stream);
        }
    }

    template void radiusMatchHamming_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    //template void radiusMatchHamming_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    template void radiusMatchHamming_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    //template void radiusMatchHamming_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
    template void radiusMatchHamming_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& nMatches, const DevMem2D& distance, hipStream_t stream);
}}}
