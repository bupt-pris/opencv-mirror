#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"

namespace cv { namespace gpu { namespace device 
{
    namespace imgproc 
    {
        template <class SrcPtr, typename D> __global__ void pyrUp(const SrcPtr src, DevMem2D_<D> dst)
        {
            typedef typename SrcPtr::elem_type src_t;
            typedef typename TypeVec<float, VecTraits<D>::cn>::vec_type sum_t;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            __shared__ sum_t s_srcPatch[10][10];
            __shared__ sum_t s_dstPatch[20][16];

            if (threadIdx.x < 10 && threadIdx.y < 10)
            {
                const int srcx = static_cast<int>((blockIdx.x * blockDim.x) / 2 + threadIdx.x) - 1;
                const int srcy = static_cast<int>((blockIdx.y * blockDim.y) / 2 + threadIdx.y) - 1;

                s_srcPatch[threadIdx.y][threadIdx.x] = saturate_cast<sum_t>(src(srcy, srcx));
            }

            __syncthreads();

            sum_t sum = VecTraits<sum_t>::all(0);

            const int evenFlag = static_cast<int>((threadIdx.x & 1) == 0);
            const int oddFlag  = static_cast<int>((threadIdx.x & 1) != 0);
            const bool eveny = ((threadIdx.y & 1) == 0);
            const int tidx = threadIdx.x;

            if (eveny)
            {
                sum = sum + (evenFlag * 0.0625f) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx - 2) >> 1)];
                sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx - 1) >> 1)];
                sum = sum + (evenFlag * 0.375f ) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx    ) >> 1)];
                sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx + 1) >> 1)];
                sum = sum + (evenFlag * 0.0625f) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx + 2) >> 1)];
            }

            s_dstPatch[2 + threadIdx.y][threadIdx.x] = sum;

            if (threadIdx.y < 2)
            {
                sum = VecTraits<sum_t>::all(0);

                if (eveny)
                {
                    sum = sum + (evenFlag * 0.0625f) * s_srcPatch[0][1 + ((tidx - 2) >> 1)];
                    sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[0][1 + ((tidx - 1) >> 1)];
                    sum = sum + (evenFlag * 0.375f ) * s_srcPatch[0][1 + ((tidx    ) >> 1)];
                    sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[0][1 + ((tidx + 1) >> 1)];
                    sum = sum + (evenFlag * 0.0625f) * s_srcPatch[0][1 + ((tidx + 2) >> 1)];
                }

                s_dstPatch[threadIdx.y][threadIdx.x] = sum;
            }

            if (threadIdx.y > 13)
            {
                sum = VecTraits<sum_t>::all(0);

                if (eveny)
                {
                    sum = sum + (evenFlag * 0.0625f) * s_srcPatch[9][1 + ((tidx - 2) >> 1)];
                    sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[9][1 + ((tidx - 1) >> 1)];
                    sum = sum + (evenFlag * 0.375f ) * s_srcPatch[9][1 + ((tidx    ) >> 1)];
                    sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[9][1 + ((tidx + 1) >> 1)];
                    sum = sum + (evenFlag * 0.0625f) * s_srcPatch[9][1 + ((tidx + 2) >> 1)];
                }

                s_dstPatch[4 + threadIdx.y][threadIdx.x] = sum;
            }

            __syncthreads();

            sum = VecTraits<sum_t>::all(0);

            const int tidy = threadIdx.y;

            sum = sum + 0.0625f * s_dstPatch[2 + tidy - 2][threadIdx.x];
            sum = sum + 0.25f   * s_dstPatch[2 + tidy - 1][threadIdx.x];
            sum = sum + 0.375f  * s_dstPatch[2 + tidy    ][threadIdx.x];
            sum = sum + 0.25f   * s_dstPatch[2 + tidy + 1][threadIdx.x];
            sum = sum + 0.0625f * s_dstPatch[2 + tidy + 2][threadIdx.x];

            if (x < dst.cols && y < dst.rows)
                dst(y, x) = saturate_cast<D>(4.0f * sum);
        }

        template <typename T, template <typename> class B> void pyrUp_caller(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, hipStream_t stream)
        {
            const dim3 block(16, 16);
            const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

            B<T> b(src.rows, src.cols);
            BorderReader< PtrStep<T>, B<T> > srcReader(src, b);

            pyrUp<<<grid, block, 0, stream>>>(srcReader, dst);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <typename T, int cn> void pyrUp_gpu(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream)
        {
            typedef typename TypeVec<T, cn>::vec_type type;

            typedef void (*caller_t)(const DevMem2D_<type>& src, const DevMem2D_<type>& dst, hipStream_t stream);

            static const caller_t callers[] = 
            {
                pyrUp_caller<type, BrdReflect101>, pyrUp_caller<type, BrdReplicate>, pyrUp_caller<type, BrdConstant>, pyrUp_caller<type, BrdReflect>, pyrUp_caller<type, BrdWrap>
            };

            callers[borderType](static_cast< DevMem2D_<type> >(src), static_cast< DevMem2D_<type> >(dst), stream);
        }

        template void pyrUp_gpu<uchar, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<uchar, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<uchar, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<uchar, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

        template void pyrUp_gpu<schar, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<schar, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<schar, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<schar, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

        template void pyrUp_gpu<ushort, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<ushort, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<ushort, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<ushort, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

        template void pyrUp_gpu<short, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<short, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<short, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<short, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

        template void pyrUp_gpu<int, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<int, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<int, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<int, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);

        template void pyrUp_gpu<float, 1>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<float, 2>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<float, 3>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
        template void pyrUp_gpu<float, 4>(const DevMem2Db& src, const DevMem2Db& dst, int borderType, hipStream_t stream);
    } // namespace imgproc
}}} // namespace cv { namespace gpu { namespace device
