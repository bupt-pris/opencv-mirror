#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/filters.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

/////////////////////////////////// Remap ///////////////////////////////////////////////
namespace cv { namespace gpu { namespace imgproc
{
    template <typename Ptr2D, typename T> __global__ void remap(const Ptr2D src, const PtrStepf mapx, const PtrStepf mapy, DevMem2D_<T> dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < dst.cols && y < dst.rows)
        {
            const float xcoo = mapx.ptr(y)[x];
            const float ycoo = mapy.ptr(y)[x];

            dst.ptr(y)[x] = saturate_cast<T>(src(ycoo, xcoo));
        }
    }
    
    template <template <typename> class Filter, template <typename> class B, typename T> struct RemapDispatcherNonStream
    {
        static void call(const DevMem2D_<T>& src, const DevMem2Df& mapx, const DevMem2Df& mapy, const DevMem2D_<T>& dst, const float* borderValue)
        {
            typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type; 
            
            dim3 block(32, 8);
            dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

            B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
            BorderReader< PtrStep_<T>, B<work_type> > brdSrc(src, brd);
            Filter< BorderReader< PtrStep_<T>, B<work_type> > > filter_src(brdSrc);

            remap<<<grid, block>>>(filter_src, mapx, mapy, dst);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }
    };

#define OPENCV_GPU_IMPLEMENT_REMAP_TEX(type) \
    texture< type , hipTextureType2D> tex_remap_ ## type (0, hipFilterModePoint, hipAddressModeClamp); \
    struct tex_remap_ ## type ## _reader \
    { \
        typedef type elem_type; \
        typedef int index_type; \
        __device__ __forceinline__ elem_type operator ()(index_type y, index_type x) const \
        { \
            return tex2D(tex_remap_ ## type , x, y); \
        } \
    }; \
    template <template <typename> class Filter, template <typename> class B> struct RemapDispatcherNonStream<Filter, B, type> \
    { \
        static void call(const DevMem2D_< type >& src, const DevMem2Df& mapx, const DevMem2Df& mapy, const DevMem2D_< type >& dst, const float* borderValue) \
        { \
            typedef typename TypeVec<float, VecTraits< type >::cn>::vec_type work_type; \
            dim3 block(32, 8); \
            dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y)); \
            TextureBinder texHandler(&tex_remap_ ## type , src); \
            tex_remap_ ## type ##_reader texSrc; \
            B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue)); \
            BorderReader< tex_remap_ ## type ##_reader, B<work_type> > brdSrc(texSrc, brd); \
            Filter< BorderReader< tex_remap_ ## type ##_reader, B<work_type> > > filter_src(brdSrc); \
            remap<<<grid, block>>>(filter_src, mapx, mapy, dst); \
            cudaSafeCall( hipGetLastError() ); \
            cudaSafeCall( hipDeviceSynchronize() ); \
        } \
    }; \
    template <template <typename> class Filter> struct RemapDispatcherNonStream<Filter, BrdReplicate, type> \
    { \
        static void call(const DevMem2D_< type >& src, const DevMem2Df& mapx, const DevMem2Df& mapy, const DevMem2D_< type >& dst, const float*) \
        { \
            dim3 block(32, 8); \
            dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y)); \
            TextureBinder texHandler(&tex_remap_ ## type , src); \
            tex_remap_ ## type ##_reader texSrc; \
            Filter< tex_remap_ ## type ##_reader > filter_src(texSrc); \
            remap<<<grid, block>>>(filter_src, mapx, mapy, dst); \
            cudaSafeCall( hipGetLastError() ); \
            cudaSafeCall( hipDeviceSynchronize() ); \
        } \
    };
    
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(uchar)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(uchar2)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(uchar4)
    
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(schar)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(char2)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(char4)
    
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(ushort)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(ushort2)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(ushort4)
    
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(short)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(short2)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(short4)
    
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(int)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(int2)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(int4)
    
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(float)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(float2)
    OPENCV_GPU_IMPLEMENT_REMAP_TEX(float4)
    
#undef OPENCV_GPU_IMPLEMENT_REMAP_TEX

    template <template <typename> class Filter, template <typename> class B, typename T> struct RemapDispatcher
    { 
        static void call(const DevMem2D_<T>& src, const DevMem2Df& mapx, const DevMem2Df& mapy, const DevMem2D_<T>& dst, const float* borderValue, hipStream_t stream)
        {
            if (stream == 0)
                RemapDispatcherNonStream<Filter, B, T>::call(src, mapx, mapy, dst, borderValue);
            else
                callStream(src, mapx, mapy, dst, borderValue, stream);
        }
        
        static void callStream(const DevMem2D_<T>& src, const DevMem2Df& mapx, const DevMem2Df& mapy, const DevMem2D_<T>& dst, const float* borderValue, hipStream_t stream)
        {
            typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type; 
            
            dim3 block(32, 8);
            dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

            B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
            BorderReader< PtrStep_<T>, B<work_type> > brd_src(src, brd);
            Filter< BorderReader< PtrStep_<T>, B<work_type> > > filter_src(brd_src);

            remap<<<grid, block, 0, stream>>>(filter_src, mapx, mapy, dst);
            cudaSafeCall( hipGetLastError() );
        }
    };

    template <typename T> void remap_gpu(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2D_<T>& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D_<T>& dst, const float* borderValue, hipStream_t stream);

        static const caller_t callers[3][5] = 
        {
            { 
                RemapDispatcher<PointFilter, BrdReflect101, T>::call, 
                RemapDispatcher<PointFilter, BrdReplicate, T>::call, 
                RemapDispatcher<PointFilter, BrdConstant, T>::call, 
                RemapDispatcher<PointFilter, BrdReflect, T>::call, 
                RemapDispatcher<PointFilter, BrdWrap, T>::call 
            },
            { 
                RemapDispatcher<LinearFilter, BrdReflect101, T>::call, 
                RemapDispatcher<LinearFilter, BrdReplicate, T>::call, 
                RemapDispatcher<LinearFilter, BrdConstant, T>::call, 
                RemapDispatcher<LinearFilter, BrdReflect, T>::call, 
                RemapDispatcher<LinearFilter, BrdWrap, T>::call 
            },
            { 
                RemapDispatcher<CubicFilter, BrdReflect101, T>::call, 
                RemapDispatcher<CubicFilter, BrdReplicate, T>::call, 
                RemapDispatcher<CubicFilter, BrdConstant, T>::call, 
                RemapDispatcher<CubicFilter, BrdReflect, T>::call, 
                RemapDispatcher<CubicFilter, BrdWrap, T>::call 
            }
        };

        callers[interpolation][borderMode](static_cast< DevMem2D_<T> >(src), xmap, ymap, static_cast< DevMem2D_<T> >(dst), borderValue, stream);
    }

    template void remap_gpu<uchar >(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<uchar2>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<uchar3>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<uchar4>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    
    template void remap_gpu<schar>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<char2>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<char3>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<char4>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    
    template void remap_gpu<ushort >(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<ushort2>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<ushort3>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<ushort4>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    
    template void remap_gpu<short >(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<short2>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<short3>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<short4>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    
    template void remap_gpu<uint >(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<uint2>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<uint3>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<uint4>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    
    template void remap_gpu<int >(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<int2>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<int3>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<int4>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    
    template void remap_gpu<float >(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<float2>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<float3>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);
    template void remap_gpu<float4>(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, const DevMem2D& dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream);

/////////////////////////////////// MeanShiftfiltering ///////////////////////////////////////////////

    texture<uchar4, 2> tex_meanshift;

    __device__ short2 do_mean_shift(int x0, int y0, unsigned char* out, 
                                    size_t out_step, int cols, int rows, 
                                    int sp, int sr, int maxIter, float eps)
    {
        int isr2 = sr*sr;
        uchar4 c = tex2D(tex_meanshift, x0, y0 );

        // iterate meanshift procedure
        for( int iter = 0; iter < maxIter; iter++ )
        {
            int count = 0;
            int s0 = 0, s1 = 0, s2 = 0, sx = 0, sy = 0;
            float icount;

            //mean shift: process pixels in window (p-sigmaSp)x(p+sigmaSp)
            int minx = x0-sp;
            int miny = y0-sp;
            int maxx = x0+sp;
            int maxy = y0+sp;

            for( int y = miny; y <= maxy; y++)
            {
                int rowCount = 0;
                for( int x = minx; x <= maxx; x++ )
                {                    
                    uchar4 t = tex2D( tex_meanshift, x, y );

                    int norm2 = (t.x - c.x) * (t.x - c.x) + (t.y - c.y) * (t.y - c.y) + (t.z - c.z) * (t.z - c.z);
                    if( norm2 <= isr2 )
                    {
                        s0 += t.x; s1 += t.y; s2 += t.z;
                        sx += x; rowCount++;
                    }
                }
                count += rowCount;
                sy += y*rowCount;
            }

            if( count == 0 )
                break;

            icount = 1.f/count;
            int x1 = __float2int_rz(sx*icount);
            int y1 = __float2int_rz(sy*icount);
            s0 = __float2int_rz(s0*icount);
            s1 = __float2int_rz(s1*icount);
            s2 = __float2int_rz(s2*icount);

            int norm2 = (s0 - c.x) * (s0 - c.x) + (s1 - c.y) * (s1 - c.y) + (s2 - c.z) * (s2 - c.z);

            bool stopFlag = (x0 == x1 && y0 == y1) || (abs(x1-x0) + abs(y1-y0) + norm2 <= eps);

            x0 = x1; y0 = y1;
            c.x = s0; c.y = s1; c.z = s2;

            if( stopFlag )
                break;
        }

        int base = (blockIdx.y * blockDim.y + threadIdx.y) * out_step + (blockIdx.x * blockDim.x + threadIdx.x) * 4 * sizeof(uchar);
        *(uchar4*)(out + base) = c;

        return make_short2((short)x0, (short)y0);
    }

    extern "C" __global__ void meanshift_kernel( unsigned char* out, size_t out_step, int cols, int rows, 
                                                 int sp, int sr, int maxIter, float eps )
    {
        int x0 = blockIdx.x * blockDim.x + threadIdx.x;
        int y0 = blockIdx.y * blockDim.y + threadIdx.y;

        if( x0 < cols && y0 < rows )
            do_mean_shift(x0, y0, out, out_step, cols, rows, sp, sr, maxIter, eps);
    }

    extern "C" __global__ void meanshiftproc_kernel( unsigned char* outr, size_t outrstep, 
                                                 unsigned char* outsp, size_t outspstep, 
                                                 int cols, int rows, 
                                                 int sp, int sr, int maxIter, float eps )
    {
        int x0 = blockIdx.x * blockDim.x + threadIdx.x;
        int y0 = blockIdx.y * blockDim.y + threadIdx.y;

        if( x0 < cols && y0 < rows )
        {            
            int basesp = (blockIdx.y * blockDim.y + threadIdx.y) * outspstep + (blockIdx.x * blockDim.x + threadIdx.x) * 2 * sizeof(short);
            *(short2*)(outsp + basesp) = do_mean_shift(x0, y0, outr, outrstep, cols, rows, sp, sr, maxIter, eps);
        }
    }

    extern "C" void meanShiftFiltering_gpu(const DevMem2D& src, DevMem2D dst, int sp, int sr, int maxIter, float eps)
    {
        dim3 grid(1, 1, 1);
        dim3 threads(32, 8, 1);
        grid.x = divUp(src.cols, threads.x);
        grid.y = divUp(src.rows, threads.y);

        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
        cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

        meanshift_kernel<<< grid, threads >>>( dst.data, dst.step, dst.cols, dst.rows, sp, sr, maxIter, eps );
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
        cudaSafeCall( hipUnbindTexture( tex_meanshift ) );        
    }
    extern "C" void meanShiftProc_gpu(const DevMem2D& src, DevMem2D dstr, DevMem2D dstsp, int sp, int sr, int maxIter, float eps) 
    {
        dim3 grid(1, 1, 1);
        dim3 threads(32, 8, 1);
        grid.x = divUp(src.cols, threads.x);
        grid.y = divUp(src.rows, threads.y);

        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
        cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

        meanshiftproc_kernel<<< grid, threads >>>( dstr.data, dstr.step, dstsp.data, dstsp.step, dstr.cols, dstr.rows, sp, sr, maxIter, eps );
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
        cudaSafeCall( hipUnbindTexture( tex_meanshift ) );        
    }

/////////////////////////////////// drawColorDisp ///////////////////////////////////////////////

    template <typename T>
    __device__ unsigned int cvtPixel(T d, int ndisp, float S = 1, float V = 1)
    {        
        unsigned int H = ((ndisp-d) * 240)/ndisp;

        unsigned int hi = (H/60) % 6;
        float f = H/60.f - H/60;
        float p = V * (1 - S);
        float q = V * (1 - f * S);
        float t = V * (1 - (1 - f) * S);

        float3 res;
        
        if (hi == 0) //R = V,	G = t,	B = p
        {
            res.x = p;
            res.y = t;
            res.z = V;
        }

        if (hi == 1) // R = q,	G = V,	B = p
        {
            res.x = p;
            res.y = V;
            res.z = q;
        }        
        
        if (hi == 2) // R = p,	G = V,	B = t
        {
            res.x = t;
            res.y = V;
            res.z = p;
        }
            
        if (hi == 3) // R = p,	G = q,	B = V
        {
            res.x = V;
            res.y = q;
            res.z = p;
        }

        if (hi == 4) // R = t,	G = p,	B = V
        {
            res.x = V;
            res.y = p;
            res.z = t;
        }

        if (hi == 5) // R = V,	G = p,	B = q
        {
            res.x = q;
            res.y = p;
            res.z = V;
        }
        const unsigned int b = (unsigned int)(max(0.f, min (res.x, 1.f)) * 255.f);
        const unsigned int g = (unsigned int)(max(0.f, min (res.y, 1.f)) * 255.f);
        const unsigned int r = (unsigned int)(max(0.f, min (res.z, 1.f)) * 255.f);
        const unsigned int a = 255U;

        return (a << 24) + (r << 16) + (g << 8) + b;    
    } 

    __global__ void drawColorDisp(uchar* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) 
        {
            uchar4 d4 = *(uchar4*)(disp + y * disp_step + x);

            uint4 res;
            res.x = cvtPixel(d4.x, ndisp);
            res.y = cvtPixel(d4.y, ndisp);
            res.z = cvtPixel(d4.z, ndisp);
            res.w = cvtPixel(d4.w, ndisp);
                    
            uint4* line = (uint4*)(out_image + y * out_step);
            line[x >> 2] = res;
        }
    }

    __global__ void drawColorDisp(short* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) 
        {
            short2 d2 = *(short2*)(disp + y * disp_step + x);

            uint2 res;
            res.x = cvtPixel(d2.x, ndisp);            
            res.y = cvtPixel(d2.y, ndisp);

            uint2* line = (uint2*)(out_image + y * out_step);
            line[x >> 1] = res;
        }
    }


    void drawColorDisp_gpu(const DevMem2D& src, const DevMem2D& dst, int ndisp, const hipStream_t& stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x << 2);
        grid.y = divUp(src.rows, threads.y);
         
        drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step, dst.data, dst.step, src.cols, src.rows, ndisp);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() ); 
    }

    void drawColorDisp_gpu(const DevMem2D_<short>& src, const DevMem2D& dst, int ndisp, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x << 1);
        grid.y = divUp(src.rows, threads.y);
         
        drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step / sizeof(short), dst.data, dst.step, src.cols, src.rows, ndisp);
        cudaSafeCall( hipGetLastError() );
        
        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

/////////////////////////////////// reprojectImageTo3D ///////////////////////////////////////////////

    __constant__ float cq[16];

    template <typename T>
    __global__ void reprojectImageTo3D(const T* disp, size_t disp_step, float* xyzw, size_t xyzw_step, int rows, int cols)
    {        
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y < rows && x < cols)
        {

            float qx = cq[1] * y + cq[3], qy = cq[5] * y + cq[7];
            float qz = cq[9] * y + cq[11], qw = cq[13] * y + cq[15];

            qx += x * cq[0]; 
            qy += x * cq[4];
            qz += x * cq[8];
            qw += x * cq[12];

            T d = *(disp + disp_step * y + x);

            float iW = 1.f / (qw + cq[14] * d);
            float4 v;
            v.x = (qx + cq[2] * d) * iW;
            v.y = (qy + cq[6] * d) * iW;
            v.z = (qz + cq[10] * d) * iW;
            v.w = 1.f;

            *(float4*)(xyzw + xyzw_step * y + (x * 4)) = v;
        }
    }

    template <typename T>
    inline void reprojectImageTo3D_caller(const DevMem2D_<T>& disp, const DevMem2Df& xyzw, const float* q, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(disp.cols, threads.x);
        grid.y = divUp(disp.rows, threads.y);

        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cq), q, 16 * sizeof(float)) );

        reprojectImageTo3D<<<grid, threads, 0, stream>>>(disp.data, disp.step / sizeof(T), xyzw.data, xyzw.step / sizeof(float), disp.rows, disp.cols);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void reprojectImageTo3D_gpu(const DevMem2D& disp, const DevMem2Df& xyzw, const float* q, const hipStream_t& stream)
    {
        reprojectImageTo3D_caller(disp, xyzw, q, stream);
    }

    void reprojectImageTo3D_gpu(const DevMem2D_<short>& disp, const DevMem2Df& xyzw, const float* q, const hipStream_t& stream)
    {
        reprojectImageTo3D_caller(disp, xyzw, q, stream);
    }

//////////////////////////////////////// Extract Cov Data ////////////////////////////////////////////////

    __global__ void extractCovData_kernel(const int cols, const int rows, const PtrStepf Dx, 
                                          const PtrStepf Dy, PtrStepf dst)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {            
            float dx = Dx.ptr(y)[x];
            float dy = Dy.ptr(y)[x];

            dst.ptr(y)[x] = dx * dx;
            dst.ptr(y + rows)[x] = dx * dy;
            dst.ptr(y + (rows << 1))[x] = dy * dy;
        }
    }

    void extractCovData_caller(const DevMem2Df Dx, const DevMem2Df Dy, PtrStepf dst)
    {
        dim3 threads(32, 8);
        dim3 grid(divUp(Dx.cols, threads.x), divUp(Dx.rows, threads.y));

        extractCovData_kernel<<<grid, threads>>>(Dx.cols, Dx.rows, Dx, Dy, dst);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }

/////////////////////////////////////////// Corner Harris /////////////////////////////////////////////////

    texture<float, 2> harrisDxTex;
    texture<float, 2> harrisDyTex;

    __global__ void cornerHarris_kernel(const int cols, const int rows, const int block_size, const float k,
                                        PtrStep dst)
    {
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            float a = 0.f;
            float b = 0.f;
            float c = 0.f;

            const int ibegin = y - (block_size / 2);
            const int jbegin = x - (block_size / 2);
            const int iend = ibegin + block_size;
            const int jend = jbegin + block_size;

            for (int i = ibegin; i < iend; ++i)
            {
                for (int j = jbegin; j < jend; ++j)
                {
                    float dx = tex2D(harrisDxTex, j, i);
                    float dy = tex2D(harrisDyTex, j, i);
                    a += dx * dx;
                    b += dx * dy;
                    c += dy * dy;
                }
            }

            ((float*)dst.ptr(y))[x] = a * c - b * b - k * (a + c) * (a + c);
        }
    }

    template <typename BR, typename BC>
    __global__ void cornerHarris_kernel(const int cols, const int rows, const int block_size, const float k,
                                        PtrStep dst, BR border_row, BC border_col)
    {
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            float a = 0.f;
            float b = 0.f;
            float c = 0.f;

            const int ibegin = y - (block_size / 2);
            const int jbegin = x - (block_size / 2);
            const int iend = ibegin + block_size;
            const int jend = jbegin + block_size;

            for (int i = ibegin; i < iend; ++i)
            {
                int y = border_col.idx_row(i);
                for (int j = jbegin; j < jend; ++j)
                {
                    int x = border_row.idx_col(j);
                    float dx = tex2D(harrisDxTex, x, y);
                    float dy = tex2D(harrisDyTex, x, y);
                    a += dx * dx;
                    b += dx * dy;
                    c += dy * dy;
                }
            }

            ((float*)dst.ptr(y))[x] = a * c - b * b - k * (a + c) * (a + c);
        }
    }

    void cornerHarris_caller(const int block_size, const float k, const DevMem2D Dx, const DevMem2D Dy, DevMem2D dst, 
                             int border_type)
    {
        const int rows = Dx.rows;
        const int cols = Dx.cols;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
        hipBindTexture2D(0, harrisDxTex, Dx.data, desc, Dx.cols, Dx.rows, Dx.step);
        hipBindTexture2D(0, harrisDyTex, Dy.data, desc, Dy.cols, Dy.rows, Dy.step);
        harrisDxTex.filterMode = hipFilterModePoint;
        harrisDyTex.filterMode = hipFilterModePoint;

        switch (border_type) 
        {
        case BORDER_REFLECT101_GPU:
            cornerHarris_kernel<<<grid, threads>>>(
                    cols, rows, block_size, k, dst, BrdRowReflect101<void>(cols), BrdColReflect101<void>(rows));
            break;
        case BORDER_REPLICATE_GPU:
            harrisDxTex.addressMode[0] = hipAddressModeClamp;
            harrisDxTex.addressMode[1] = hipAddressModeClamp;
            harrisDyTex.addressMode[0] = hipAddressModeClamp;
            harrisDyTex.addressMode[1] = hipAddressModeClamp;
            cornerHarris_kernel<<<grid, threads>>>(cols, rows, block_size, k, dst);
            break;
        }

        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );

        cudaSafeCall(hipUnbindTexture(harrisDxTex));
        cudaSafeCall(hipUnbindTexture(harrisDyTex));
    }

/////////////////////////////////////////// Corner Min Eigen Val /////////////////////////////////////////////////

    texture<float, 2> minEigenValDxTex;
    texture<float, 2> minEigenValDyTex;

    __global__ void cornerMinEigenVal_kernel(const int cols, const int rows, const int block_size, 
                                             PtrStep dst)
    {
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            float a = 0.f;
            float b = 0.f;
            float c = 0.f;

            const int ibegin = y - (block_size / 2);
            const int jbegin = x - (block_size / 2);
            const int iend = ibegin + block_size;
            const int jend = jbegin + block_size;

            for (int i = ibegin; i < iend; ++i)
            {
                for (int j = jbegin; j < jend; ++j)
                {
                    float dx = tex2D(minEigenValDxTex, j, i);
                    float dy = tex2D(minEigenValDyTex, j, i);
                    a += dx * dx;
                    b += dx * dy;
                    c += dy * dy;
                }
            }

            a *= 0.5f;
            c *= 0.5f;
            ((float*)dst.ptr(y))[x] = (a + c) - sqrtf((a - c) * (a - c) + b * b);
        }
    }


    template <typename BR, typename BC>
    __global__ void cornerMinEigenVal_kernel(const int cols, const int rows, const int block_size, 
                                             PtrStep dst, BR border_row, BC border_col)
    {
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            float a = 0.f;
            float b = 0.f;
            float c = 0.f;

            const int ibegin = y - (block_size / 2);
            const int jbegin = x - (block_size / 2);
            const int iend = ibegin + block_size;
            const int jend = jbegin + block_size;

            for (int i = ibegin; i < iend; ++i)
            {
                int y = border_col.idx_row(i);
                for (int j = jbegin; j < jend; ++j)
                {
                    int x = border_row.idx_col(j);
                    float dx = tex2D(minEigenValDxTex, x, y);
                    float dy = tex2D(minEigenValDyTex, x, y);
                    a += dx * dx;
                    b += dx * dy;
                    c += dy * dy;
                }
            }

            a *= 0.5f;
            c *= 0.5f;
            ((float*)dst.ptr(y))[x] = (a + c) - sqrtf((a - c) * (a - c) + b * b);
        }
    }

    void cornerMinEigenVal_caller(const int block_size, const DevMem2D Dx, const DevMem2D Dy, DevMem2D dst,
                                  int border_type)
    {
        const int rows = Dx.rows;
        const int cols = Dx.cols;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
        hipBindTexture2D(0, minEigenValDxTex, Dx.data, desc, Dx.cols, Dx.rows, Dx.step);
        hipBindTexture2D(0, minEigenValDyTex, Dy.data, desc, Dy.cols, Dy.rows, Dy.step);
        minEigenValDxTex.filterMode = hipFilterModePoint;
        minEigenValDyTex.filterMode = hipFilterModePoint;

        switch (border_type)
        {
        case BORDER_REFLECT101_GPU:
            cornerMinEigenVal_kernel<<<grid, threads>>>(
                    cols, rows, block_size, dst, BrdRowReflect101<void>(cols), BrdColReflect101<void>(rows));
            break;
        case BORDER_REPLICATE_GPU:
            minEigenValDxTex.addressMode[0] = hipAddressModeClamp;
            minEigenValDxTex.addressMode[1] = hipAddressModeClamp;
            minEigenValDyTex.addressMode[0] = hipAddressModeClamp;
            minEigenValDyTex.addressMode[1] = hipAddressModeClamp;
            cornerMinEigenVal_kernel<<<grid, threads>>>(cols, rows, block_size, dst);
            break;
        }

        cudaSafeCall( hipGetLastError() );

        cudaSafeCall(hipDeviceSynchronize());

        cudaSafeCall(hipUnbindTexture(minEigenValDxTex));
        cudaSafeCall(hipUnbindTexture(minEigenValDyTex));
    }

////////////////////////////// Column Sum //////////////////////////////////////

    __global__ void column_sumKernel_32F(int cols, int rows, const PtrStep src, const PtrStep dst)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;

        if (x < cols)
        {
            const unsigned char* src_data = src.data + x * sizeof(float);
            unsigned char* dst_data = dst.data + x * sizeof(float);

            float sum = 0.f;
            for (int y = 0; y < rows; ++y)
            {
                sum += *(const float*)src_data;
                *(float*)dst_data = sum;
                src_data += src.step;
                dst_data += dst.step;
            }
        }
    }


    void columnSum_32F(const DevMem2D src, const DevMem2D dst)
    {
        dim3 threads(256);
        dim3 grid(divUp(src.cols, threads.x));

        column_sumKernel_32F<<<grid, threads>>>(src.cols, src.rows, src, dst);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }


    //////////////////////////////////////////////////////////////////////////
    // mulSpectrums

    __global__ void mulSpectrumsKernel(const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b, 
                                       DevMem2D_<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;    
        const int y = blockIdx.y * blockDim.y + threadIdx.y;    

        if (x < c.cols && y < c.rows) 
        {
            c.ptr(y)[x] = hipCmulf(a.ptr(y)[x], b.ptr(y)[x]);
        }
    }


    void mulSpectrums(const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b, 
                      DevMem2D_<hipfftComplex> c)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulSpectrumsKernel<<<grid, threads>>>(a, b, c);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }


    //////////////////////////////////////////////////////////////////////////
    // mulSpectrums_CONJ

    __global__ void mulSpectrumsKernel_CONJ(
            const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b,
            DevMem2D_<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;    
        const int y = blockIdx.y * blockDim.y + threadIdx.y;    

        if (x < c.cols && y < c.rows) 
        {
            c.ptr(y)[x] = hipCmulf(a.ptr(y)[x], hipConjf(b.ptr(y)[x]));
        }
    }


    void mulSpectrums_CONJ(const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b, 
                           DevMem2D_<hipfftComplex> c)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulSpectrumsKernel_CONJ<<<grid, threads>>>(a, b, c);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }


    //////////////////////////////////////////////////////////////////////////
    // mulAndScaleSpectrums

    __global__ void mulAndScaleSpectrumsKernel(
            const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b, 
            float scale, DevMem2D_<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < c.cols && y < c.rows) 
        {
            hipfftComplex v = hipCmulf(a.ptr(y)[x], b.ptr(y)[x]);
            c.ptr(y)[x] = make_hipFloatComplex(hipCrealf(v) * scale, hipCimagf(v) * scale);
        }
    }


    void mulAndScaleSpectrums(const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b,
                              float scale, DevMem2D_<hipfftComplex> c)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulAndScaleSpectrumsKernel<<<grid, threads>>>(a, b, scale, c);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }


    //////////////////////////////////////////////////////////////////////////
    // mulAndScaleSpectrums_CONJ

    __global__ void mulAndScaleSpectrumsKernel_CONJ(
            const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b,
            float scale, DevMem2D_<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < c.cols && y < c.rows) 
        {
            hipfftComplex v = hipCmulf(a.ptr(y)[x], hipConjf(b.ptr(y)[x]));
            c.ptr(y)[x] = make_hipFloatComplex(hipCrealf(v) * scale, hipCimagf(v) * scale);
        }
    }


    void mulAndScaleSpectrums_CONJ(const PtrStep_<hipfftComplex> a, const PtrStep_<hipfftComplex> b,
                                  float scale, DevMem2D_<hipfftComplex> c)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulAndScaleSpectrumsKernel_CONJ<<<grid, threads>>>(a, b, scale, c);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }


    /////////////////////////////////////////////////////////////////////////
    // downsample

    template <typename T, int cn>
    __global__ void downsampleKernel(const PtrStep_<T> src, DevMem2D_<T> dst)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < dst.cols && y < dst.rows)
        {
            int ch_x = x / cn;
            dst.ptr(y)[x] = src.ptr(y*2)[ch_x*2*cn + x - ch_x*cn];
        }
    }


    template <typename T, int cn>
    void downsampleCaller(const DevMem2D src, DevMem2D dst, hipStream_t stream)
    {
        dim3 threads(32, 8);
        dim3 grid(divUp(dst.cols, threads.x), divUp(dst.rows, threads.y));

        downsampleKernel<T,cn><<<grid, threads, 0, stream>>>(DevMem2D_<T>(src), DevMem2D_<T>(dst));
        cudaSafeCall(hipGetLastError());
        
        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    template void downsampleCaller<uchar,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<uchar,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<uchar,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<uchar,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<short,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<short,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<short,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<short,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<float,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<float,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<float,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void downsampleCaller<float,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);


    //////////////////////////////////////////////////////////////////////////
    // upsample

    template <typename T, int cn>
    __global__ void upsampleKernel(const PtrStep_<T> src, DevMem2D_<T> dst)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < dst.cols && y < dst.rows)
        {
            int ch_x = x / cn;
            T val = ((ch_x & 1) || (y & 1)) ? 0 : src.ptr(y/2)[ch_x/2*cn + x - ch_x*cn];
            dst.ptr(y)[x] = val;
        }
    }


    template <typename T, int cn>
    void upsampleCaller(const DevMem2D src, DevMem2D dst, hipStream_t stream)
    {
        dim3 threads(32, 8);
        dim3 grid(divUp(dst.cols, threads.x), divUp(dst.rows, threads.y));

        upsampleKernel<T,cn><<<grid, threads, 0, stream>>>(DevMem2D_<T>(src), DevMem2D_<T>(dst));
        cudaSafeCall(hipGetLastError());

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    template void upsampleCaller<uchar,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<uchar,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<uchar,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<uchar,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<short,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<short,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<short,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<short,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<float,1>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<float,2>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<float,3>(const DevMem2D src, DevMem2D dst, hipStream_t stream);
    template void upsampleCaller<float,4>(const DevMem2D src, DevMem2D dst, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // pyrDown

    template <typename T, typename B> __global__ void pyrDown(const PtrStep_<T> src, PtrStep_<T> dst, const B b, int dst_cols)
    {
        typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type value_type;

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y;

        __shared__ value_type smem[256 + 4];

        value_type sum;
        
        const int src_y = 2*y;

        sum = VecTraits<value_type>::all(0);
        
        sum = sum + 0.0625f * b.at(src_y - 2, x, src.data, src.step);
        sum = sum + 0.25f   * b.at(src_y - 1, x, src.data, src.step);
        sum = sum + 0.375f  * b.at(src_y    , x, src.data, src.step);
        sum = sum + 0.25f   * b.at(src_y + 1, x, src.data, src.step);
        sum = sum + 0.0625f * b.at(src_y + 2, x, src.data, src.step);

        smem[2 + threadIdx.x] = sum;

        if (threadIdx.x < 2)
        {
            const int left_x = x - 2 + threadIdx.x;

            sum = VecTraits<value_type>::all(0);
        
            sum = sum + 0.0625f * b.at(src_y - 2, left_x, src.data, src.step);
            sum = sum + 0.25f   * b.at(src_y - 1, left_x, src.data, src.step);
            sum = sum + 0.375f  * b.at(src_y    , left_x, src.data, src.step);
            sum = sum + 0.25f   * b.at(src_y + 1, left_x, src.data, src.step);
            sum = sum + 0.0625f * b.at(src_y + 2, left_x, src.data, src.step);

            smem[threadIdx.x] = sum;
        }

        if (threadIdx.x > 253)
        {
            const int right_x = x + threadIdx.x + 2;

            sum = VecTraits<value_type>::all(0);
        
            sum = sum + 0.0625f * b.at(src_y - 2, right_x, src.data, src.step);
            sum = sum + 0.25f   * b.at(src_y - 1, right_x, src.data, src.step);
            sum = sum + 0.375f  * b.at(src_y    , right_x, src.data, src.step);
            sum = sum + 0.25f   * b.at(src_y + 1, right_x, src.data, src.step);
            sum = sum + 0.0625f * b.at(src_y + 2, right_x, src.data, src.step);

            smem[4 + threadIdx.x] = sum;
        }

        __syncthreads();

        if (threadIdx.x < 128)
        {
            const int tid2 = threadIdx.x * 2;

            sum = VecTraits<value_type>::all(0);

            sum = sum + 0.0625f * smem[2 + tid2 - 2];
            sum = sum + 0.25f   * smem[2 + tid2 - 1];
            sum = sum + 0.375f  * smem[2 + tid2    ];
            sum = sum + 0.25f   * smem[2 + tid2 + 1];
            sum = sum + 0.0625f * smem[2 + tid2 + 2];

            const int dst_x = (blockIdx.x * blockDim.x + tid2) / 2;

            if (dst_x < dst_cols)
                dst.ptr(y)[dst_x] = saturate_cast<T>(sum);
        }
    }

    template <typename T, template <typename> class B> void pyrDown_caller(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        const dim3 block(256);
        const dim3 grid(divUp(src.cols, block.x), dst.rows);

        B<T> b(src.rows, src.cols);

        pyrDown<T><<<grid, block, 0, stream>>>(src, dst, b, dst.cols);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <typename T, int cn> void pyrDown_gpu(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream)
    {
        typedef typename TypeVec<T, cn>::vec_type type;

        typedef void (*caller_t)(const DevMem2D_<type>& src, const DevMem2D_<type>& dst, hipStream_t stream);

        static const caller_t callers[] = 
        {
            pyrDown_caller<type, BrdReflect101>, pyrDown_caller<type, BrdReplicate>, pyrDown_caller<type, BrdConstant>, pyrDown_caller<type, BrdReflect>, pyrDown_caller<type, BrdWrap>
        };

        callers[borderType](static_cast< DevMem2D_<type> >(src), static_cast< DevMem2D_<type> >(dst), stream);
    }

    template void pyrDown_gpu<uchar, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<uchar, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<uchar, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<uchar, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    template void pyrDown_gpu<schar, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<schar, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<schar, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<schar, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    template void pyrDown_gpu<ushort, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<ushort, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<ushort, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<ushort, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    template void pyrDown_gpu<short, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<short, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<short, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<short, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    template void pyrDown_gpu<int, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<int, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<int, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<int, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    template void pyrDown_gpu<float, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<float, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<float, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrDown_gpu<float, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // pyrUp

    template <typename T, typename B> __global__ void pyrUp(const PtrStep_<T> src, DevMem2D_<T> dst, const B b)
    {
        typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type value_type;

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        __shared__ T smem1[10][10];
        __shared__ value_type smem2[20][16];

        value_type sum;

        if (threadIdx.x < 10 && threadIdx.y < 10)
            smem1[threadIdx.y][threadIdx.x] = b.at(blockIdx.y * blockDim.y / 2 + threadIdx.y - 1, blockIdx.x * blockDim.x / 2 + threadIdx.x - 1, src.data, src.step);

        __syncthreads();

        const int tidx = threadIdx.x;

        sum = VecTraits<value_type>::all(0);

        sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[1 + threadIdx.y / 2][1 + ((tidx - 2) >> 1)];
        sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[1 + threadIdx.y / 2][1 + ((tidx - 1) >> 1)];
        sum = sum + (tidx % 2 == 0) * 0.375f  * smem1[1 + threadIdx.y / 2][1 + ((tidx    ) >> 1)];
        sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[1 + threadIdx.y / 2][1 + ((tidx + 1) >> 1)];
        sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[1 + threadIdx.y / 2][1 + ((tidx + 2) >> 1)];

        smem2[2 + threadIdx.y][tidx] = sum;

        if (threadIdx.y < 2)
        {
            sum = VecTraits<value_type>::all(0);

            sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[0][1 + ((tidx - 2) >> 1)];
            sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[0][1 + ((tidx - 1) >> 1)];
            sum = sum + (tidx % 2 == 0) * 0.375f  * smem1[0][1 + ((tidx    ) >> 1)];
            sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[0][1 + ((tidx + 1) >> 1)];
            sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[0][1 + ((tidx + 2) >> 1)];

            smem2[threadIdx.y][tidx] = sum;
        }

        if (threadIdx.y > 13)
        {
            sum = VecTraits<value_type>::all(0);

            sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[9][1 + ((tidx - 2) >> 1)];
            sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[9][1 + ((tidx - 1) >> 1)];
            sum = sum + (tidx % 2 == 0) * 0.375f  * smem1[9][1 + ((tidx    ) >> 1)];
            sum = sum + (tidx % 2 != 0) * 0.25f   * smem1[9][1 + ((tidx + 1) >> 1)];
            sum = sum + (tidx % 2 == 0) * 0.0625f * smem1[9][1 + ((tidx + 2) >> 1)];

            smem2[4 + threadIdx.y][tidx] = sum;
        }

        __syncthreads();

        sum = VecTraits<value_type>::all(0);

        sum = sum + (tidx % 2 == 0) * 0.0625f * smem2[2 + threadIdx.y - 2][tidx];
        sum = sum + (tidx % 2 != 0) * 0.25f   * smem2[2 + threadIdx.y - 1][tidx];
        sum = sum + (tidx % 2 == 0) * 0.375f  * smem2[2 + threadIdx.y    ][tidx];
        sum = sum + (tidx % 2 != 0) * 0.25f   * smem2[2 + threadIdx.y + 1][tidx];
        sum = sum + (tidx % 2 == 0) * 0.0625f * smem2[2 + threadIdx.y + 2][tidx];

        if (x < dst.cols && y < dst.rows)
            dst.ptr(y)[x] = saturate_cast<T>(4.0f * sum);
    }

    template <typename T, template <typename> class B> void pyrUp_caller(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        const dim3 block(16, 16);
        const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

        B<T> b(src.rows, src.cols);

        pyrUp<T><<<grid, block, 0, stream>>>(src, dst, b);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <typename T, int cn> void pyrUp_gpu(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream)
    {
        typedef typename TypeVec<T, cn>::vec_type type;

        typedef void (*caller_t)(const DevMem2D_<type>& src, const DevMem2D_<type>& dst, hipStream_t stream);

        static const caller_t callers[] = 
        {
            pyrUp_caller<type, BrdReflect101>, pyrUp_caller<type, BrdReplicate>, pyrUp_caller<type, BrdConstant>, pyrUp_caller<type, BrdReflect>, pyrUp_caller<type, BrdWrap>
        };

        callers[borderType](static_cast< DevMem2D_<type> >(src), static_cast< DevMem2D_<type> >(dst), stream);
    }

    template void pyrUp_gpu<uchar, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<uchar, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<uchar, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<uchar, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    template void pyrUp_gpu<schar, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<schar, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<schar, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<schar, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    template void pyrUp_gpu<ushort, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<ushort, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<ushort, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<ushort, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    template void pyrUp_gpu<short, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<short, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<short, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<short, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    template void pyrUp_gpu<int, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<int, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<int, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<int, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    template void pyrUp_gpu<float, 1>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<float, 2>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<float, 3>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);
    template void pyrUp_gpu<float, 4>(const DevMem2D& src, const DevMem2D& dst, int borderType, hipStream_t stream);

    //////////////////////////////////////////////////////////////////////////
    // buildWarpMaps

    namespace build_warp_maps
    {

        __constant__ float cr[9];
        __constant__ float crinv[9];
        __constant__ float cf, cs;
        __constant__ float chalf_w, chalf_h;
        __constant__ float cdist;
    }


    class PlaneMapper
    {
    public:
        static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
        {
            using namespace build_warp_maps;

            float x_ = u / cs;
            float y_ = v / cs;

            float z;
            x = crinv[0]*x_ + crinv[1]*y_ + crinv[2]*cdist;
            y = crinv[3]*x_ + crinv[4]*y_ + crinv[5]*cdist;
            z = crinv[6]*x_ + crinv[7]*y_ + crinv[8]*cdist;

            x = cf*x/z + chalf_w;
            y = cf*y/z + chalf_h;
        }
    };


    class CylindricalMapper
    {
    public:
        static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
        {
            using namespace build_warp_maps;

            u /= cs;
            float x_ = sinf(u);
            float y_ = v / cs;
            float z_ = cosf(u);

            float z;
            x = crinv[0]*x_ + crinv[1]*y_ + crinv[2]*z_;
            y = crinv[3]*x_ + crinv[4]*y_ + crinv[5]*z_;
            z = crinv[6]*x_ + crinv[7]*y_ + crinv[8]*z_;

            x = cf*x/z + chalf_w;
            y = cf*y/z + chalf_h;
        }
    };


    class SphericalMapper
    {
    public:
        static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
        {
            using namespace build_warp_maps;

            v /= cs;
            u /= cs;

            float sinv = sinf(v);
            float x_ = sinv * sinf(u);
            float y_ = -cosf(v);
            float z_ = sinv * cosf(u);

            float z;
            x = crinv[0]*x_ + crinv[1]*y_ + crinv[2]*z_;
            y = crinv[3]*x_ + crinv[4]*y_ + crinv[5]*z_;
            z = crinv[6]*x_ + crinv[7]*y_ + crinv[8]*z_;

            x = cf*x/z + chalf_w;
            y = cf*y/z + chalf_h;
        }
    };


    template <typename Mapper>
    __global__ void buildWarpMapsKernel(int tl_u, int tl_v, int cols, int rows,
                                        PtrStepf map_x, PtrStepf map_y)
    {
        int du = blockIdx.x * blockDim.x + threadIdx.x;
        int dv = blockIdx.y * blockDim.y + threadIdx.y;
        if (du < cols && dv < rows)
        {
            float u = tl_u + du;
            float v = tl_v + dv;
            float x, y;
            Mapper::mapBackward(u, v, x, y);
            map_x.ptr(dv)[du] = x;
            map_y.ptr(dv)[du] = y;
        }
    }


    void buildWarpPlaneMaps(int tl_u, int tl_v, DevMem2Df map_x, DevMem2Df map_y,
                            const float r[9], const float rinv[9], float f, float s, float dist,
                            float half_w, float half_h, hipStream_t stream)
    {
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr), r, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::crinv), rinv, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cf), &f, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cs), &s, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_w), &half_w, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_h), &half_h, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cdist), &dist, sizeof(float)));

        int cols = map_x.cols;
        int rows = map_x.rows;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        buildWarpMapsKernel<PlaneMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
        cudaSafeCall(hipGetLastError());
        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    void buildWarpCylindricalMaps(int tl_u, int tl_v, DevMem2Df map_x, DevMem2Df map_y,
                                  const float r[9], const float rinv[9], float f, float s,
                                  float half_w, float half_h, hipStream_t stream)
    {
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr), r, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::crinv), rinv, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cf), &f, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cs), &s, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_w), &half_w, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_h), &half_h, sizeof(float)));

        int cols = map_x.cols;
        int rows = map_x.rows;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        buildWarpMapsKernel<CylindricalMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
        cudaSafeCall(hipGetLastError());
        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    void buildWarpSphericalMaps(int tl_u, int tl_v, DevMem2Df map_x, DevMem2Df map_y,
                                const float r[9], const float rinv[9], float f, float s,
                                float half_w, float half_h, hipStream_t stream)
    {
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr), r, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::crinv), rinv, 9*sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cf), &f, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cs), &s, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_w), &half_w, sizeof(float)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::chalf_h), &half_h, sizeof(float)));

        int cols = map_x.cols;
        int rows = map_x.rows;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        buildWarpMapsKernel<SphericalMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
        cudaSafeCall(hipGetLastError());
        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


}}}


