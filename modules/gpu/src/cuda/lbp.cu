#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <opencv2/gpu/device/lbp.hpp>
#include <opencv2/gpu/device/vec_traits.hpp>
#include <opencv2/gpu/device/saturate_cast.hpp>

namespace cv { namespace gpu { namespace device
{
    namespace lbp
    {

        texture<int, hipTextureType2D, hipReadModeElementType> tintegral(false, hipFilterModePoint, hipAddressModeClamp);

        struct LBP
        {
            __host__ __device__ __forceinline__ LBP(const LBP& other) {(void)other;}
            __host__ __device__ __forceinline__ LBP() {}

            __device__ __forceinline__ int operator() (int ty, int tx, int fh, int fw, int& shift) const
            {
                int anchors[9];

                anchors[0]  = tex2D(tintegral, tx, ty);
                anchors[1]  = tex2D(tintegral, tx + fw, ty);
                anchors[0] -= anchors[1];
                anchors[2]  = tex2D(tintegral, tx + fw * 2, ty);
                anchors[1] -= anchors[2];
                anchors[2] -= tex2D(tintegral, tx + fw * 3, ty);

                ty += fh;
                anchors[3]  = tex2D(tintegral, tx, ty);
                anchors[4]  = tex2D(tintegral, tx + fw, ty);
                anchors[3] -= anchors[4];
                anchors[5]  = tex2D(tintegral, tx + fw * 2, ty);
                anchors[4] -= anchors[5];
                anchors[5] -= tex2D(tintegral, tx + fw * 3, ty);

                anchors[0] -= anchors[3];
                anchors[1] -= anchors[4];
                anchors[2] -= anchors[5];
                // 0 - 2 contains s0 - s2

                ty += fh;
                anchors[6]  = tex2D(tintegral, tx, ty);
                anchors[7]  = tex2D(tintegral, tx + fw, ty);
                anchors[6] -= anchors[7];
                anchors[8]  = tex2D(tintegral, tx + fw * 2, ty);
                anchors[7] -= anchors[8];
                anchors[8] -= tex2D(tintegral, tx + fw * 3, ty);

                anchors[3] -= anchors[6];
                anchors[4] -= anchors[7];
                anchors[5] -= anchors[8];
                // 3 - 5 contains s3 - s5

                anchors[0] -= anchors[4];
                anchors[1] -= anchors[4];
                anchors[2] -= anchors[4];
                anchors[3] -= anchors[4];
                anchors[5] -= anchors[4];

                int response = (~(anchors[0] >> 31)) & 4;
                response |= (~(anchors[1] >> 31)) & 2;;
                response |= (~(anchors[2] >> 31)) & 1;

                shift = (~(anchors[5] >> 31)) & 16;
                shift |= (~(anchors[3] >> 31)) & 1;

                ty += fh;
                anchors[0]  = tex2D(tintegral, tx, ty);
                anchors[1]  = tex2D(tintegral, tx + fw, ty);
                anchors[0] -= anchors[1];
                anchors[2]  = tex2D(tintegral, tx + fw * 2, ty);
                anchors[1] -= anchors[2];
                anchors[2] -= tex2D(tintegral, tx + fw * 3, ty);

                anchors[6] -= anchors[0];
                anchors[7] -= anchors[1];
                anchors[8] -= anchors[2];
                // 0 -2 contains s6 - s8

                anchors[6] -= anchors[4];
                anchors[7] -= anchors[4];
                anchors[8] -= anchors[4];

                shift |= (~(anchors[6] >> 31)) & 2;
                shift |= (~(anchors[7] >> 31)) & 4;
                shift |= (~(anchors[8] >> 31)) & 8;
                return response;
            }
        };

        void bindIntegral(DevMem2Di integral)
        {
            hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
            cudaSafeCall( hipBindTexture2D(0, &tintegral, integral.ptr(), &desc, (size_t)integral.cols, (size_t)integral.rows, (size_t)integral.step));
        }

        void unbindIntegral()
        {
             cudaSafeCall( hipUnbindTexture(&tintegral));
        }

        struct Classifier
        {
            __host__ __device__ __forceinline__ Classifier(const Stage* _stages, const ClNode* _nodes, const float* _leaves, const int* _subsets, const uchar4* _features,
                const int _nstages, const int _clWidth, const int _clHeight, const float _scale, const int _step, const int _subsetSize)
            : stages(_stages), nodes(_nodes), leaves(_leaves), subsets(_subsets), features(_features), nstages(_nstages), clWidth(_clWidth), clHeight(_clHeight),
              scale(_scale), step(_step), subsetSize(_subsetSize){}

            __device__ __forceinline__ void operator() (int y, int x, DevMem2D_<int4> objects, const unsigned int maxN, unsigned int* n) const
            {
                int current_node = 0;
                int current_leave = 0;

                for (int s = 0; s < nstages; ++s)
                {
                    float sum = 0;
                    Stage stage = stages[s];
                    for (int t = 0; t < stage.ntrees; t++)
                    {
                        ClNode node = nodes[current_node];
                        uchar4 feature = features[node.featureIdx];

                        int shift;
                        int c = evaluator(y + feature.y, x + feature.x, feature.w, feature.z, shift);
                        int idx =  (subsets[ current_node * subsetSize + c] & ( 1 << shift)) ? current_leave : current_leave + 1;
                        sum += leaves[idx];

                        current_node += 1;
                        current_leave += 2;
                    }

                    if (sum < stage.threshold)
                        return;
                }

                int4 rect;
                rect.x = roundf(x * scale);
                rect.y = roundf(y * scale);
                rect.z = clWidth;
                rect.w = clHeight;

#if defined (__CUDA_ARCH__) && (__CUDA_ARCH__ < 120)
                int res = __atomicInc(n, maxN);
#else
                int res = atomicInc(n, maxN);
#endif
                objects(0, res) = rect;
            }

            const Stage*  stages;
            const ClNode* nodes;
            const float* leaves;
            const int* subsets;
            const uchar4* features;

            const int nstages;
            const int clWidth;
            const int clHeight;
            const float scale;
            const int step;
            const int subsetSize;
            const LBP evaluator;
        };

        __global__ void lbp_classify_stump(const Classifier classifier, DevMem2D_<int4> objects, const unsigned int maxN, unsigned int* n)
        {
            int x = threadIdx.x * classifier.step;
            int y = blockIdx.x * classifier.step;

            classifier(y, x, objects, maxN, n);
        }

        __global__ void lbp_classify_stump(const Classifier classifier, DevMem2D_<int4> objects, const unsigned int maxN, unsigned int* n, int maxX)
        {
            int ftid = blockIdx.x * blockDim.x + threadIdx.x;
            int y = ftid / maxX;
            int x = ftid - y * maxX;

            classifier(y * classifier.step, x * classifier.step, objects, maxN, n);
        }

        template<typename Pr>
        __global__ void disjoin(int4* candidates, int4* objects, unsigned int n, int groupThreshold, float grouping_eps, unsigned int* nclasses)
        {
            unsigned int tid = threadIdx.x;
            extern __shared__ int sbuff[];

            int* labels = sbuff;
            int* rrects = (int*)(sbuff + n);

            Pr predicate(grouping_eps);
            partition(candidates, n, labels, predicate);

            rrects[tid * 4 + 0] = 0;
            rrects[tid * 4 + 1] = 0;
            rrects[tid * 4 + 2] = 0;
            rrects[tid * 4 + 3] = 0;
            __syncthreads();

            int cls = labels[tid];
#if defined (__CUDA_ARCH__) && (__CUDA_ARCH__ < 120)
            __atomicAdd((rrects + cls * 4 + 0), candidates[tid].x);
            __atomicAdd((rrects + cls * 4 + 1), candidates[tid].y);
            __atomicAdd((rrects + cls * 4 + 2), candidates[tid].z);
            __atomicAdd((rrects + cls * 4 + 3), candidates[tid].w);
#else
            atomicAdd((rrects + cls * 4 + 0), candidates[tid].x);
            atomicAdd((rrects + cls * 4 + 1), candidates[tid].y);
            atomicAdd((rrects + cls * 4 + 2), candidates[tid].z);
            atomicAdd((rrects + cls * 4 + 3), candidates[tid].w);
#endif
            __syncthreads();
            labels[tid] = 0;

            __syncthreads();
#if defined (__CUDA_ARCH__) && (__CUDA_ARCH__ < 120)
            __atomicInc((unsigned int*)labels + cls, n);
#else
            atomicInc((unsigned int*)labels + cls, n);
#endif
            __syncthreads();
            *nclasses = 0;

            int active = labels[tid];
            if (active)
            {
                int* r1 = rrects + tid * 4;
                float s = 1.f / active;
                r1[0] = saturate_cast<int>(r1[0] * s);
                r1[1] = saturate_cast<int>(r1[1] * s);
                r1[2] = saturate_cast<int>(r1[2] * s);
                r1[3] = saturate_cast<int>(r1[3] * s);
            }
            __syncthreads();

            if (active && active >= groupThreshold)
            {
                int* r1 = rrects + tid * 4;
                int4 r_out;
                r_out.x = r1[0];
                r_out.y = r1[1];
                r_out.z = r1[2];
                r_out.w = r1[3];
#if defined (__CUDA_ARCH__) && (__CUDA_ARCH__ < 120)
                objects[__atomicInc(nclasses, n)] = r_out;
#else
                int aidx = atomicInc(nclasses, n);
                objects[aidx] = r_out;
#endif
            }
        }

        void classifyStump(const DevMem2Db& mstages, const int nstages, const DevMem2Di& mnodes, const DevMem2Df& mleaves, const DevMem2Di& msubsets, const DevMem2Db& mfeatures,
                           const int workWidth, const int workHeight, const int clWidth, const int clHeight, float scale, int step, int subsetSize, DevMem2D_<int4> objects, unsigned int* classified)
        {
            int blocks  = ceilf(workHeight / (float)step);
            int threads = ceilf(workWidth / (float)step);

            Classifier clr((Stage*)(mstages.ptr()), (ClNode*)(mnodes.ptr()), mleaves.ptr(), msubsets.ptr(), (uchar4*)(mfeatures.ptr()), nstages, clWidth, clHeight, scale, step, subsetSize);
            lbp_classify_stump<<<blocks, threads>>>(clr,  objects, objects.cols, classified);
        }

        void classifyStumpFixed(const DevMem2Db& mstages, const int nstages, const DevMem2Di& mnodes, const DevMem2Df& mleaves, const DevMem2Di& msubsets, const DevMem2Db& mfeatures,
                           const int workWidth, const int workHeight, const int clWidth, const int clHeight, float scale, int step, int subsetSize, DevMem2D_<int4> objects, unsigned int* classified)
        {
            const int THREADS_BLOCK = 256;
            int work_amount = ceilf(workHeight / (float)step) * ceilf(workWidth / (float)step);
            int blocks  = divUp(work_amount, THREADS_BLOCK);

            Classifier clr((Stage*)(mstages.ptr()), (ClNode*)(mnodes.ptr()), mleaves.ptr(), msubsets.ptr(), (uchar4*)(mfeatures.ptr()), nstages, clWidth, clHeight, scale, step, subsetSize);
            lbp_classify_stump<<<blocks, THREADS_BLOCK>>>(clr, objects, objects.cols, classified, workWidth >> 1);
        }

        int connectedConmonents(DevMem2D_<int4> candidates, int ncandidates, DevMem2D_<int4> objects, int groupThreshold, float grouping_eps, unsigned int* nclasses)
        {
            int threads = ncandidates;
            int smem_amount = threads * sizeof(int) + threads * sizeof(int4);
            disjoin<InSameComponint><<<1, threads, smem_amount>>>((int4*)candidates.ptr(), (int4*)objects.ptr(), ncandidates, groupThreshold, grouping_eps, nclasses);
            return 0;
        }
    }
}}}