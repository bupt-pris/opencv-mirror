#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/vec_distance.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace cv { namespace gpu { namespace bfmatcher
{
    template <typename VecDiff, typename Dist, typename T, typename Mask>
    __device__ void distanceCalcLoop(const PtrStep_<T>& query, const DevMem2D_<T>& train, const Mask& m, int queryIdx,
        typename Dist::result_type& distMin1, typename Dist::result_type& distMin2, int& bestTrainIdx1, int& bestTrainIdx2, 
        typename Dist::result_type* smem)
    {
        const VecDiff vecDiff(query.ptr(queryIdx), train.cols, (typename Dist::value_type*)smem, threadIdx.y * blockDim.x + threadIdx.x, threadIdx.x);
        
        typename Dist::result_type* sdiffRow = smem + blockDim.x * threadIdx.y;
        
        distMin1 = numeric_limits<typename Dist::result_type>::max();
        distMin2 = numeric_limits<typename Dist::result_type>::max();

        bestTrainIdx1 = -1;
        bestTrainIdx2 = -1;

        for (int trainIdx = threadIdx.y; trainIdx < train.rows; trainIdx += blockDim.y)
        {
            if (m(queryIdx, trainIdx))
            {
                Dist dist;

                const T* trainRow = train.ptr(trainIdx);
                
                vecDiff.calc(trainRow, train.cols, dist, sdiffRow, threadIdx.x);

                const typename Dist::result_type val = dist;

                if (val < distMin1)
                {
                    distMin1 = val;
                    bestTrainIdx1 = trainIdx;
                }
                else if (val < distMin2)
                {
                    distMin2 = val;
                    bestTrainIdx2 = trainIdx;
                }
            }
        }
    }

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename VecDiff, typename Dist, typename T, typename Mask>
    __global__ void knnMatch2(const PtrStep_<T> query, const DevMem2D_<T> train, const Mask m, int2* trainIdx, float2* distance)
    {
        typedef typename Dist::result_type result_type;
        typedef typename Dist::value_type value_type;

        __shared__ result_type smem[BLOCK_DIM_X * BLOCK_DIM_Y];

        const int queryIdx = blockIdx.x;

        result_type distMin1;
        result_type distMin2;

        int bestTrainIdx1;
        int bestTrainIdx2;

        distanceCalcLoop<VecDiff, Dist>(query, train, m, queryIdx, distMin1, distMin2, bestTrainIdx1, bestTrainIdx2, smem);
        __syncthreads();

        volatile result_type* sdistMinRow = smem;
        volatile int* sbestTrainIdxRow = (int*)(sdistMinRow + 2 * BLOCK_DIM_Y);

        if (threadIdx.x == 0)
        {
            sdistMinRow[threadIdx.y] = distMin1;
            sdistMinRow[threadIdx.y + BLOCK_DIM_Y] = distMin2;

            sbestTrainIdxRow[threadIdx.y] = bestTrainIdx1;            
            sbestTrainIdxRow[threadIdx.y + BLOCK_DIM_Y] = bestTrainIdx2;
        }
        __syncthreads();

        if (threadIdx.x == 0 && threadIdx.y == 0)
        {
            distMin1 = numeric_limits<result_type>::max();
            distMin2 = numeric_limits<result_type>::max();

            bestTrainIdx1 = -1;
            bestTrainIdx2 = -1;

            #pragma unroll
            for (int i = 0; i < BLOCK_DIM_Y; ++i)
            {
                result_type val = sdistMinRow[i];

                if (val < distMin1)
                {
                    distMin1 = val;
                    bestTrainIdx1 = sbestTrainIdxRow[i];
                }
                else if (val < distMin2)
                {
                    distMin2 = val;
                    bestTrainIdx2 = sbestTrainIdxRow[i];
                }
            }

            #pragma unroll
            for (int i = BLOCK_DIM_Y; i < 2 * BLOCK_DIM_Y; ++i)
            {
                result_type val = sdistMinRow[i];

                if (val < distMin2)
                {
                    distMin2 = val;
                    bestTrainIdx2 = sbestTrainIdxRow[i];
                }
            }

            trainIdx[queryIdx] = make_int2(bestTrainIdx1, bestTrainIdx2);
            distance[queryIdx] = make_float2(distMin1, distMin2);
        }
    }

    ///////////////////////////////////////////////////////////////////////////////
    // Knn 2 Match kernel caller

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename Dist, typename T, typename Mask>
    void knnMatch2Simple_caller(const DevMem2D_<T>& query, const DevMem2D_<T>& train, const Mask& mask, 
        const DevMem2D_<int2>& trainIdx, const DevMem2D_<float2>& distance, 
        hipStream_t stream)
    {
        const dim3 grid(query.rows, 1, 1);
        const dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y, 1);

        knnMatch2<BLOCK_DIM_X, BLOCK_DIM_Y, VecDiffGlobal<BLOCK_DIM_X, T>, Dist, T>
            <<<grid, threads, 0, stream>>>(query, train, mask, trainIdx, distance);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, int MAX_LEN, bool LEN_EQ_MAX_LEN, typename Dist, typename T, typename Mask>
    void knnMatch2Cached_caller(const DevMem2D_<T>& query, const DevMem2D_<T>& train, const Mask& mask, 
        const DevMem2D_<int2>& trainIdx, const DevMem2D_<float2>& distance, 
        hipStream_t stream)
    {
        StaticAssert<BLOCK_DIM_X * BLOCK_DIM_Y >= MAX_LEN>::check(); // block size must be greter than descriptors length
        StaticAssert<MAX_LEN % BLOCK_DIM_X == 0>::check();           // max descriptors length must divide to blockDimX

        const dim3 grid(query.rows, 1, 1);
        const dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y, 1);

        knnMatch2<BLOCK_DIM_X, BLOCK_DIM_Y, VecDiffCachedRegister<BLOCK_DIM_X, MAX_LEN, LEN_EQ_MAX_LEN, typename Dist::value_type>, Dist, T>
              <<<grid, threads, 0, stream>>>(query, train, mask, trainIdx.data, distance.data);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    ///////////////////////////////////////////////////////////////////////////////
    // Knn 2 Match Dispatcher
    
    template <typename Dist, typename T, typename Mask>
    void knnMatch2Dispatcher(const DevMem2D_<T>& query, const DevMem2D_<T>& train, const Mask& mask, 
        const DevMem2D& trainIdx, const DevMem2D& distance, 
        int cc, hipStream_t stream)
    {
        if (query.cols < 64)
        {
            knnMatch2Cached_caller<16, 16, 64, false, Dist>(
                query, train, mask, 
                static_cast< DevMem2D_<int2> >(trainIdx), static_cast< DevMem2D_<float2> >(distance),
                stream);
        }
        else if (query.cols == 64)
        {
            knnMatch2Cached_caller<16, 16, 64, true, Dist>(
                query, train, mask, 
                static_cast< DevMem2D_<int2> >(trainIdx), static_cast< DevMem2D_<float2> >(distance), 
                stream);
        }
        else if (query.cols < 128)
        {
            knnMatch2Cached_caller<16, 16, 128, false, Dist>(
                query, train, mask, 
                static_cast< DevMem2D_<int2> >(trainIdx), static_cast< DevMem2D_<float2> >(distance), 
                stream);
        }
        else if (query.cols == 128 && cc >= 12)
        {
            knnMatch2Cached_caller<16, 16, 128, true, Dist>(
                query, train, mask, 
                static_cast< DevMem2D_<int2> >(trainIdx), static_cast< DevMem2D_<float2> >(distance), 
                stream);
        }
        else if (query.cols < 256 && cc >= 12)
        {
            knnMatch2Cached_caller<16, 16, 256, false, Dist>(
                query, train, mask, 
                static_cast< DevMem2D_<int2> >(trainIdx), static_cast< DevMem2D_<float2> >(distance), 
                stream);
        }
        else if (query.cols == 256 && cc >= 12)
        {
            knnMatch2Cached_caller<16, 16, 256, true, Dist>(
                query, train, mask, 
                static_cast< DevMem2D_<int2> >(trainIdx), static_cast< DevMem2D_<float2> >(distance), 
                stream);
        }
        else
        {
            knnMatch2Simple_caller<16, 16, Dist>(
                query, train, mask, 
                static_cast< DevMem2D_<int2> >(trainIdx), static_cast< DevMem2D_<float2> >(distance),
                stream);
        }
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // Calc distance kernel

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename Dist, typename T, typename Mask>
    __global__ void calcDistance(const PtrStep_<T> query, const DevMem2D_<T> train, const Mask mask, PtrStepf distance)
    {
        __shared__ typename Dist::result_type sdiff[BLOCK_DIM_X * BLOCK_DIM_Y];

        typename Dist::result_type* sdiff_row = sdiff + BLOCK_DIM_X * threadIdx.y;
        
        const int queryIdx = blockIdx.x;
        const T* queryDescs = query.ptr(queryIdx);

        const int trainIdx = blockIdx.y * BLOCK_DIM_Y + threadIdx.y;

        if (trainIdx < train.rows)
        {
            const T* trainDescs = train.ptr(trainIdx);

            typename Dist::result_type myDist = numeric_limits<typename Dist::result_type>::max();

            if (mask(queryIdx, trainIdx))
            {
                Dist dist;

                calcVecDiffGlobal<BLOCK_DIM_X>(queryDescs, trainDescs, train.cols, dist, sdiff_row, threadIdx.x);

                myDist = dist;
            }
            
            if (threadIdx.x == 0)
                distance.ptr(queryIdx)[trainIdx] = myDist;
        }
    }

    ///////////////////////////////////////////////////////////////////////////////
    // Calc distance kernel caller

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename Dist, typename T, typename Mask>
    void calcDistance_caller(const DevMem2D_<T>& query, const DevMem2D_<T>& train, const Mask& mask, const DevMem2Df& distance, hipStream_t stream)
    {
        const dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
        const dim3 grid(query.rows, divUp(train.rows, BLOCK_DIM_Y), 1);

        calcDistance<BLOCK_DIM_X, BLOCK_DIM_Y, Dist, T><<<grid, threads, 0, stream>>>(query, train, mask, distance);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <typename Dist, typename T, typename Mask>
    void calcDistanceDispatcher(const DevMem2D_<T>& query, const DevMem2D_<T>& train, const Mask& mask, const DevMem2D& allDist, hipStream_t stream)
    {
        calcDistance_caller<16, 16, Dist>(query, train, mask, static_cast<DevMem2Df>(allDist), stream);
    }

    ///////////////////////////////////////////////////////////////////////////////
    // find knn match kernel

    template <int BLOCK_SIZE> __global__ void findBestMatch(DevMem2Df allDist_, int i, PtrStepi trainIdx_, PtrStepf distance_)
    {
        const int SMEM_SIZE = BLOCK_SIZE > 64 ? BLOCK_SIZE : 64;
        __shared__ float sdist[SMEM_SIZE];
        __shared__ int strainIdx[SMEM_SIZE];

        const int queryIdx = blockIdx.x;

        float* allDist = allDist_.ptr(queryIdx);
        int* trainIdx = trainIdx_.ptr(queryIdx);
        float* distance = distance_.ptr(queryIdx);

        float dist = numeric_limits<float>::max();
        int bestIdx = -1;
        
        for (int i = threadIdx.x; i < allDist_.cols; i += BLOCK_SIZE)
        {
            float reg = allDist[i];
            if (reg < dist)
            {
                dist = reg;
                bestIdx = i;
            }
        }

        sdist[threadIdx.x] = dist;
        strainIdx[threadIdx.x] = bestIdx;
        __syncthreads();

        reducePredVal<BLOCK_SIZE>(sdist, dist, strainIdx, bestIdx, threadIdx.x, less<volatile float>());

        if (threadIdx.x == 0)
        {
            if (dist < numeric_limits<float>::max())
            {
                allDist[bestIdx] = numeric_limits<float>::max();
                trainIdx[i] = bestIdx;
                distance[i] = dist;
            }
        }
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // find knn match kernel caller

    template <int BLOCK_SIZE> void findKnnMatch_caller(int k, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist, hipStream_t stream)
    {
        const dim3 threads(BLOCK_SIZE, 1, 1);
        const dim3 grid(trainIdx.rows, 1, 1);

        for (int i = 0; i < k; ++i)
        {
            findBestMatch<BLOCK_SIZE><<<grid, threads, 0, stream>>>(allDist, i, trainIdx, distance);
            cudaSafeCall( hipGetLastError() );
        }

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void findKnnMatchDispatcher(int k, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, hipStream_t stream)
    {
        findKnnMatch_caller<256>(k, static_cast<DevMem2Di>(trainIdx), static_cast<DevMem2Df>(distance), static_cast<DevMem2Df>(allDist), stream);
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // knn match Dispatcher

    template <typename Dist, typename T>
    void knnMatchDispatcher(const DevMem2D_<T>& query, const DevMem2D_<T>& train, int k, const DevMem2D& mask, 
        const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, 
        int cc, hipStream_t stream)
    {
        if (mask.data)
        {
            if (k == 2)
            {
                knnMatch2Dispatcher<Dist>(query, train, SingleMask(mask), trainIdx, distance, cc, stream);
                return;
            }

            calcDistanceDispatcher<Dist>(query, train, SingleMask(mask), allDist, stream);
        }
        else
        {
            if (k == 2)
            {
                knnMatch2Dispatcher<Dist>(query, train, WithOutMask(), trainIdx, distance, cc, stream);
                return;
            }

            calcDistanceDispatcher<Dist>(query, train, WithOutMask(), allDist, stream);
        }

        findKnnMatchDispatcher(k, trainIdx, distance, allDist, stream);
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // knn match caller

    template <typename T> void knnMatchL1_gpu(const DevMem2D& query, const DevMem2D& train, int k, const DevMem2D& mask, 
        const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, 
        int cc, hipStream_t stream)
    {
        knnMatchDispatcher< L1Dist<T> >(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), k, mask, trainIdx, distance, allDist, cc, stream);
    }

    template void knnMatchL1_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    //template void knnMatchL1_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    template void knnMatchL1_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    template void knnMatchL1_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    template void knnMatchL1_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    template void knnMatchL1_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);

    template <typename T> void knnMatchL2_gpu(const DevMem2D& query, const DevMem2D& train, int k, const DevMem2D& mask, 
        const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist,
        int cc, hipStream_t stream)
    {
        knnMatchDispatcher<L2Dist>(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), k, mask, trainIdx, distance, allDist, cc, stream);
    }

    //template void knnMatchL2_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    //template void knnMatchL2_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    //template void knnMatchL2_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    //template void knnMatchL2_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    //template void knnMatchL2_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    template void knnMatchL2_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);

    template <typename T> void knnMatchHamming_gpu(const DevMem2D& query, const DevMem2D& train, int k, const DevMem2D& mask,
        const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, 
        int cc, hipStream_t stream)
    {
        knnMatchDispatcher<HammingDist>(static_cast< DevMem2D_<T> >(query), static_cast< DevMem2D_<T> >(train), k, mask, trainIdx, distance, allDist, cc, stream);
    }

    template void knnMatchHamming_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    //template void knnMatchHamming_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    template void knnMatchHamming_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    //template void knnMatchHamming_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
    template void knnMatchHamming_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int k, const DevMem2D& mask, const DevMem2D& trainIdx, const DevMem2D& distance, const DevMem2D& allDist, int cc, hipStream_t stream);
}}}
